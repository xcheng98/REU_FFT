#include "hip/hip_runtime.h"
/*
 * Implementing fft4 algorithm
 * Input is multiple fp32 vector, number given by B
 * First split every input vector to two fp16 vectors (integrate in the same file)
 * It's not a complete FFT
 * To be used recursively by gfft
 * Using unified memory
 */

// C includes
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>

// CUDA includes
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>

// Matrix and vector
#include "helper/my_vector.h"
#include "helper/my_matrix.h"
#include "helper/my_const.h"
#include "util/fp32_to_fp16.h"

// CUDA helper: to check error
#include "nvidia_helper/checkCudaErrors.h"

const float UPPER_BOUND = 1.0f;
const int BATCH = 4;
// const int BLOCKSIZE = 16;

fft::MatrixH F4_re;
fft::MatrixH F4_im;

FFT_S init_F4()
{
    F4_re.element(1, 1) = 1.0f;
    F4_re.element(2, 1) = 1.0f;
    F4_re.element(3, 1) = 1.0f;
    F4_re.element(4, 1) = 1.0f;
    F4_re.element(1, 2) = 1.0f;
    F4_re.element(2, 2) = 0.0f;
    F4_re.element(3, 2) =-1.0f;
    F4_re.element(4, 2) = 0.0f;
    F4_re.element(1, 3) = 1.0f;
    F4_re.element(2, 3) =-1.0f;
    F4_re.element(3, 3) = 1.0f;
    F4_re.element(4, 3) =-1.0f;
    F4_re.element(1, 4) = 1.0f;
    F4_re.element(2, 4) = 0.0f;
    F4_re.element(3, 4) =-1.0f;
    F4_re.element(4, 4) = 0.0f;

    F4_im.element(1, 1) = 0.0f;
    F4_im.element(2, 1) = 0.0f;
    F4_im.element(3, 1) = 0.0f;
    F4_im.element(4, 1) = 0.0f;
    F4_im.element(1, 2) = 0.0f;
    F4_im.element(2, 2) =-1.0f;
    F4_im.element(3, 2) = 0.0f;
    F4_im.element(4, 2) = 1.0f;
    F4_im.element(1, 3) = 0.0f;
    F4_im.element(2, 3) = 0.0f;
    F4_im.element(3, 3) = 0.0f;
    F4_im.element(4, 3) = 0.0f;
    F4_im.element(1, 4) = 0.0f;
    F4_im.element(2, 4) = 1.0f;
    F4_im.element(3, 4) = 0.0f;
    F4_im.element(4, 4) =-1.0f;

    return FFT_SUCCESS;
}

namespace fft{
    template <int BLOCK_SIZE> __global__ void _half_to_single(int size, half* input, float* output)
    {
        /* 
         * Convert the input half-precision vector to single-precision
         * Block and thread layout should be 1D
         * Block size need to be specified
         * */
        int bx = blockIdx.x;
        int tx = threadIdx.x;
        int index = bx * BLOCK_SIZE + tx;

        if (index < size) {
            output[index] = __half2float(input[index]);
        }
    }
}


FFT_S fft4(int B, fft::MatrixF X_re, fft::MatrixF X_im, fft::MatrixF FX_re, fft::MatrixF FX_im) 
{
    // Variable declaration
    hipblasStatus_t status;
    hipblasHandle_t handle;

    //// Unified variables
    float *scales; // = *re_s1, *re_s2, *im_s1, *im_s2;
    half *X_split; // = *X_re_hi, *X_re_lo, *X_im_hi, *X_im_lo;
    float *result1, *result2; // Store the intermediate result
    //// Scaling variables
    float alpha = 1.0f, beta = 0.0f; 

    // Initialize cublas
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return FFT_FAILURE;
    }

    status = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH); // allow Tensor Core
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS setting math mode error\n");
        return FFT_FAILURE;
    }


    //  Allocate unified memory with 0 initialization
    checkCudaErrors(hipMallocManaged((void **) &scales, B * 4 * sizeof(float)));
    checkCudaErrors(hipMemset(scales, 0.0f, B * 4 * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **) &X_split, 4 * B * 4 * sizeof(half)));
    checkCudaErrors(hipMemset(X_split, 0.0f, 4 * B * 4 * sizeof(half)));
    checkCudaErrors(hipMallocManaged((void **) &result1, 4 * B * 4 * sizeof(result1[0])));
    checkCudaErrors(hipMemset(result1, 0.0f, 4 * B * 4 * sizeof(result1[0])));
    checkCudaErrors(hipMallocManaged((void **) &result2, 4 * 4 * sizeof(result2[0])));
    checkCudaErrors(hipMemset(result2, 0.0f, 4 * 4 * sizeof(result2[0])));

    // Split input
    //// Initialize Matrix and Vector data structure to store split result
    fft::MatrixH X_re_hi;
    X_re_hi.width = B;
    X_re_hi.height = 4;
    X_re_hi.array = X_split + 4 * B * 0;

    fft::MatrixH X_re_lo;
    X_re_lo.width = B;
    X_re_lo.height = 4;
    X_re_lo.array = X_split + 4 * B * 1;

    fft::MatrixH X_im_hi;
    X_im_hi.width = B;
    X_im_hi.height = 4;
    X_im_hi.array = X_split + 4 * B * 2;

    fft::MatrixH X_im_lo;
    X_im_lo.width = B;
    X_im_lo.height = 4;
    X_im_lo.array = X_split + 4 * B * 3;

    fft::VectorF re_s1;
    re_s1.size = B;
    re_s1.array = scales + B * 0;

    fft::VectorF re_s2;
    re_s2.size = B;
    re_s2.array = scales + B * 1;

    fft::VectorF im_s1;
    im_s1.size = B;
    im_s1.array = scales + B * 2;

    fft::VectorF im_s2;
    im_s2.size = B;
    im_s2.array = scales + B * 3;

    //// Call splitting function
    FFT_S fft_status;

    fft_status = split_32_to_16(X_re, X_re_hi, X_re_lo, re_s1, re_s2, 4, B);
    if (fft_status != FFT_SUCCESS){
        fprintf(stderr, "!!!!! Data splitting error (split X_re).\n");
        return FFT_FAILURE;
    }

    fft_status = split_32_to_16(X_im, X_im_hi, X_im_lo, im_s1, im_s2, 4, B);
    if (fft_status != FFT_SUCCESS){
        fprintf(stderr, "!!!!! Data splitting error (split X_im).\n");
        return FFT_FAILURE;
    }

    
    // Call cublas function and finish Matrix multiplication calculation
    //// Call cublas gemm on F4_re
    status = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, B * 4, 4, &alpha, F4_re.array,
                        HIP_R_16F, 4, X_split, HIP_R_16F, 4, &beta, result1, HIP_R_32F, 4, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS kernel execution error (a * (c, d)).\n");
        return FFT_FAILURE;
    }

    //// Call cublas gemm on F4_im
    status = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, B * 4, 4, &alpha, F4_im.array,
                        HIP_R_16F, 4, X_split, HIP_R_16F, 4, &beta, result2, HIP_R_32F, 4, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS kernel execution error (b * (c, d)).\n");
        return FFT_FAILURE;
    }


    // Scale, combine and get result, store in the first quarter and third quarter of result1
    for (int j = 0; j < B; j++)
    {
        //// Scale FM_re * X_re_h and accumulate
        alpha = re_s1.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result1 + 4 * B * 0 + 4 * j, 1, FX_re.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_re * X_re_h and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_re * X_re_l and accumulate
        alpha = re_s2.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result1 + 4 * B * 1 + 4 * j, 1, FX_re.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_re * X_re_l and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_im * X_im_h and accumulate
        alpha = -1.0f * im_s1.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result2 + 4 * B * 2 + 4 * j, 1, FX_re.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_im * X_im_h and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_im * X_im_l and accumulate
        alpha = -1.0f * im_s2.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result2 + 4 * B * 3 + 4 * j, 1, FX_re.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_im * X_im_l and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_re * X_im_h and accumulate
        alpha = im_s1.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result1 + 4 * B * 2 + 4 * j, 1, FX_im.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_re * X_im_h and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_re * X_im_l and accumulate
        alpha = im_s2.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result1 + 4 * B * 3 + 4 * j, 1, FX_im.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_re * X_im_l and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_im * X_re_h and accumulate
        alpha = re_s1.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result2 + 4 * B * 0 + 4 * j, 1, FX_im.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_im * X_re_h and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_im * X_re_l and accumulate
        alpha = re_s2.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result2 + 4 * B * 1 + 4 * j, 1, FX_im.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_im * X_re_l and accumulate).\n");
            return FFT_FAILURE;
        }
    }

    // Deallocate unified memory
    if (hipFree(scales) != hipSuccess) {
        fprintf(stderr, "!!!! unified memory free error (free scales vector)\n");
        return FFT_FAILURE;
    }

    if (hipFree(X_split) != hipSuccess) {
        fprintf(stderr, "!!!! unified memory free error (free split result matrix)\n");
        return FFT_FAILURE;
    }

    if (hipFree(result1) != hipSuccess) {
        fprintf(stderr, "!!!! unified memory free error (free result 1 Matrix)\n");
        return FFT_FAILURE;
    }

    if (hipFree(result2) != hipSuccess) {
        fprintf(stderr, "!!!! unified memory free error (free result 2 Matrix)\n");
        return FFT_FAILURE;
    }

    return FFT_SUCCESS;
}

int main()
{
    int mem_size;

    // Allocate unified memory for Fourier Matrix
    F4_re.width = 4;
    F4_re.height = 4;
    mem_size = F4_re.width * F4_re.height * sizeof(half);
    checkCudaErrors(hipMallocManaged((void **) &(F4_re.array), mem_size));
    F4_im.width = 4;
    F4_im.height = 4;
    mem_size = F4_im.width * F4_im.height * sizeof(half);
    checkCudaErrors(hipMallocManaged((void **) &(F4_im.array), mem_size));

    FFT_S status;
    status = init_F4();
    if (status != FFT_SUCCESS){
        printf("Error in Fourier matrix initialization\n");
        exit(1);
    }

    fft::MatrixF X_re;
    X_re.height = 4;
    X_re.width = BATCH;
    mem_size = X_re.height * X_re.width * sizeof(float);
    checkCudaErrors(hipMallocManaged((void **) &(X_re.array), mem_size));

    fft::MatrixF X_im;
    X_im.height = 4;
    X_im.width = BATCH;
    mem_size = X_im.height * X_im.width * sizeof(float);
    checkCudaErrors(hipMallocManaged((void **) &(X_im.array), mem_size));

    fft::MatrixF FX_re;
    FX_re.height = 4;
    FX_re.width = BATCH;
    mem_size = FX_re.height * FX_re.width * sizeof(float);
    checkCudaErrors(hipMallocManaged((void **) &(FX_re.array), mem_size));

    fft::MatrixF FX_im;
    FX_im.height = 4;
    FX_im.width = BATCH;
    mem_size = FX_im.height * FX_im.width * sizeof(float);
    checkCudaErrors(hipMallocManaged((void **) &(FX_im.array), mem_size));

    hipDeviceSynchronize();

    // Setting input value
    srand(time(NULL));
    printf("The input is: \n");
    for (int j = 1; j <= BATCH; j++){
        printf("Vector %d: \n", j);
        for (int i = 1; i <= 4; i++){
            X_re.element(i, j) = (float)rand() / (float)(RAND_MAX) * 2 * UPPER_BOUND - UPPER_BOUND;
            X_im.element(i, j) = (float)rand() / (float)(RAND_MAX) * 2 * UPPER_BOUND - UPPER_BOUND;
            printf("X[%d] = (%.10f, %.10f) \n", i, X_re.element(i, j), X_im.element(i, j));
        }
    }

    status = fft4(BATCH, X_re, X_im, FX_re, FX_im);
    if (status != FFT_SUCCESS){
        printf("Error in running fft calculation\n");
        exit(1);
    }

    hipDeviceSynchronize();

    printf("Result: \n");
    for (int j = 1; j <= BATCH; j++){
        printf("Resulting vector %d: \n", j);
        for (int i = 1; i <= 4; i++){
            printf("FX[%d] = (%.10f, %.10f) \n", i, FX_re.element(i, j), FX_im.element(i, j));
        }
    }

    checkCudaErrors(hipFree(F4_re.array));
    checkCudaErrors(hipFree(F4_im.array));

    checkCudaErrors(hipFree(X_re.array));
    checkCudaErrors(hipFree(X_im.array));
    checkCudaErrors(hipFree(FX_re.array));
    checkCudaErrors(hipFree(FX_im.array));

    return 0;
}
