/*
 * Implementing the FFT algorithm for general input
 * Input should be fp32 vectors with size equals to the power of 4
 * Number of vectors is given by BATCH (B)
 * Recursive algorithm
 * Base case is fft4
 */

// C includes
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>

// CUDA includes
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

// Matrix and vector
#include "helper/my_vector.h"
#include "helper/my_matrix.h"
#include "helper/my_const.h"

// Utility programs
#include "util/fp32_to_fp16.h"
#include "util/fourier_matrix_4.h"
#include "util/fft4.h"

// CUDA helper: to check error
#include "nvidia_helper/checkCudaErrors.h"

const float UPPER_BOUND = 1.0f;
const int BATCH = 1;
const int SIZE = 16;

extern fft::MatrixH F4_re;
extern fft::MatrixH F4_im;

FFT_S gfft(int N, int B, fft::MatrixF X_re, fft::MatrixF X_im, fft::MatrixF FX_re, fft::MatrixF FX_im) 
{
    
    if (N == 4) {
        return fft4(B, X_re, X_im, FX_re, FX_im);
    }


    // Variable declaration
    hipblasStatus_t status;
    hipblasHandle_t handle;

    //// Unified variables
    float *scales; // = *re_s1, *re_s2, *im_s1, *im_s2;
    half *X_split; // = *X_re_hi, *X_re_lo, *X_im_hi, *X_im_lo;
    float *result1, *result2; // Store the intermediate result
    //// Scaling variables
    float alpha = 1.0f, beta = 0.0f; 

    // Initialize cublas
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return FFT_FAILURE;
    }

    status = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH); // allow Tensor Core
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS setting math mode error\n");
        return FFT_FAILURE;
    }


    //  Allocate unified memory with 0 initialization
    checkCudaErrors(hipMallocManaged((void **) &scales, B * 4 * sizeof(float)));
    checkCudaErrors(hipMemset(scales, 0.0f, B * 4 * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **) &X_split, 4 * B * 4 * sizeof(half)));
    checkCudaErrors(hipMemset(X_split, 0.0f, 4 * B * 4 * sizeof(half)));
    checkCudaErrors(hipMallocManaged((void **) &result1, 4 * B * 4 * sizeof(result1[0])));
    checkCudaErrors(hipMemset(result1, 0.0f, 4 * B * 4 * sizeof(result1[0])));
    checkCudaErrors(hipMallocManaged((void **) &result2, 4 * 4 * sizeof(result2[0])));
    checkCudaErrors(hipMemset(result2, 0.0f, 4 * 4 * sizeof(result2[0])));

    // Split input
    //// Initialize Matrix and Vector data structure to store split result
    fft::MatrixH X_re_hi;
    X_re_hi.width = B;
    X_re_hi.height = 4;
    X_re_hi.array = X_split + 4 * B * 0;

    fft::MatrixH X_re_lo;
    X_re_lo.width = B;
    X_re_lo.height = 4;
    X_re_lo.array = X_split + 4 * B * 1;

    fft::MatrixH X_im_hi;
    X_im_hi.width = B;
    X_im_hi.height = 4;
    X_im_hi.array = X_split + 4 * B * 2;

    fft::MatrixH X_im_lo;
    X_im_lo.width = B;
    X_im_lo.height = 4;
    X_im_lo.array = X_split + 4 * B * 3;

    fft::VectorF re_s1;
    re_s1.size = B;
    re_s1.array = scales + B * 0;

    fft::VectorF re_s2;
    re_s2.size = B;
    re_s2.array = scales + B * 1;

    fft::VectorF im_s1;
    im_s1.size = B;
    im_s1.array = scales + B * 2;

    fft::VectorF im_s2;
    im_s2.size = B;
    im_s2.array = scales + B * 3;

    //// Call splitting function
    FFT_S fft_status;

    fft_status = split_32_to_16(X_re, X_re_hi, X_re_lo, re_s1, re_s2, 4, B);
    if (fft_status != FFT_SUCCESS){
        fprintf(stderr, "!!!!! Data splitting error (split X_re).\n");
        return FFT_FAILURE;
    }

    fft_status = split_32_to_16(X_im, X_im_hi, X_im_lo, im_s1, im_s2, 4, B);
    if (fft_status != FFT_SUCCESS){
        fprintf(stderr, "!!!!! Data splitting error (split X_im).\n");
        return FFT_FAILURE;
    }

    
    // Call cublas function and finish Matrix multiplication calculation
    //// Call cublas gemm on F4_re
    status = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, B * 4, 4, &alpha, F4_re.array,
                        HIP_R_16F, 4, X_split, HIP_R_16F, 4, &beta, result1, HIP_R_32F, 4, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS kernel execution error (a * (c, d)).\n");
        return FFT_FAILURE;
    }

    //// Call cublas gemm on F4_im
    status = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, B * 4, 4, &alpha, F4_im.array,
                        HIP_R_16F, 4, X_split, HIP_R_16F, 4, &beta, result2, HIP_R_32F, 4, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS kernel execution error (b * (c, d)).\n");
        return FFT_FAILURE;
    }


    // Scale, combine and get result, store in the first quarter and third quarter of result1
    for (int j = 0; j < B; j++)
    {
        //// Scale FM_re * X_re_h and accumulate
        alpha = re_s1.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result1 + 4 * B * 0 + 4 * j, 1, FX_re.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_re * X_re_h and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_re * X_re_l and accumulate
        alpha = re_s2.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result1 + 4 * B * 1 + 4 * j, 1, FX_re.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_re * X_re_l and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_im * X_im_h and accumulate
        alpha = -1.0f * im_s1.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result2 + 4 * B * 2 + 4 * j, 1, FX_re.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_im * X_im_h and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_im * X_im_l and accumulate
        alpha = -1.0f * im_s2.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result2 + 4 * B * 3 + 4 * j, 1, FX_re.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_im * X_im_l and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_re * X_im_h and accumulate
        alpha = im_s1.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result1 + 4 * B * 2 + 4 * j, 1, FX_im.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_re * X_im_h and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_re * X_im_l and accumulate
        alpha = im_s2.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result1 + 4 * B * 3 + 4 * j, 1, FX_im.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_re * X_im_l and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_im * X_re_h and accumulate
        alpha = re_s1.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result2 + 4 * B * 0 + 4 * j, 1, FX_im.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_im * X_re_h and accumulate).\n");
            return FFT_FAILURE;
        }

        //// Scale FM_im * X_re_l and accumulate
        alpha = re_s2.element(j + 1);
        status = hipblasSaxpy(handle, 4, &alpha, result2 + 4 * B * 1 + 4 * j, 1, FX_im.array + 4 * j, 1);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf(stderr, "!!!! CUBLAS kernel execution error (Scale FM_im * X_re_l and accumulate).\n");
            return FFT_FAILURE;
        }
    }

    // Deallocate unified memory
    if (hipFree(scales) != hipSuccess) {
        fprintf(stderr, "!!!! unified memory free error (free scales vector)\n");
        return FFT_FAILURE;
    }

    if (hipFree(X_split) != hipSuccess) {
        fprintf(stderr, "!!!! unified memory free error (free split result matrix)\n");
        return FFT_FAILURE;
    }

    if (hipFree(result1) != hipSuccess) {
        fprintf(stderr, "!!!! unified memory free error (free result 1 Matrix)\n");
        return FFT_FAILURE;
    }

    if (hipFree(result2) != hipSuccess) {
        fprintf(stderr, "!!!! unified memory free error (free result 2 Matrix)\n");
        return FFT_FAILURE;
    }

    return FFT_SUCCESS;
}

int main()
{
    int mem_size;

    // allocate unified memory for input matrix
    fft::MatrixF input_re;
    input_re.width = BATCH;
    input_re.height = SIZE;
    mem_size = input_re.width * input_re.height * sizeof(float);
    checkCudaErrors(hipMallocManaged((void **) &(input_re.array), mem_size));
    fft::MatrixF input_im;
    input_im.width = BATCH;
    input_im.height = SIZE;
    mem_size = input_im.width * input_im.height * sizeof(float);
    checkCudaErrors(hipMallocManaged((void **) &(input_im.array), mem_size));

    // Initialize the input matrix
    srand(time(NULL));
    printf("The input is: \n");
    for (int j = 1; i <= BATCH; i++){
        printf("Vector %d: \n", j);
        for (int i = 1; j <= SIZE; j++){
            input_re.element(i, j) = (float)rand() / (float)(RAND_MAX) * 2 * UPPER_BOUND - UPPER_BOUND;
            input_im.element(i, j) = (float)rand() / (float)(RAND_MAX) * 2 * UPPER_BOUND - UPPER_BOUND;
            printf("X[%d] = (%.10f, %.10f) \n", i, X_re.element(i, j), X_im.element(i, j));
        }
        printf("\n");
    }
    
    // allocate unified memory for output matrix
    fft::MatrixF output_re;
    output_re.width = BATCH;
    output_re.height = SIZE;
    mem_size = output_re.width * output_re.height * sizeof(float);
    checkCudaErrors(hipMallocManaged((void **) &(output_re.array), mem_size));
    fft::MatrixF output_im;
    output_im.width = BATCH;
    output_im.height = SIZE;
    mem_size = output_im.width * output_im.height * sizeof(float);
    checkCudaErrors(hipMallocManaged((void **) &(output_im.array), mem_size));


    status = gfft(SIZE, BATCH, input_re, input_im, output_re, output_im);
    if (status != FFT_SUCCESS){
        printf("Error in running fft algorithm\n");
        exit(1);
    }

    hipDeviceSynchronize();

    printf("Result: \n");
    for (int j = 1; j <= BATCH; j++){
        printf("Resulting vector %d: \n", j);
        for (int i = 1; i <= SIZE; i++){
            printf("FX[%d] = (%.10f, %.10f) \n", i, FX_re.element(i, j), FX_im.element(i, j));
        }
    }

    checkCudaErrors(hipFree(input_re.array));
    checkCudaErrors(hipFree(input_im.array));
    checkCudaErrors(hipFree(output_re.array));
    checkCudaErrors(hipFree(output_im.array));

    return 0;
}
