#include "hip/hip_runtime.h"

      PadData(h_signal, &h_padded_signal, SIGNAL_SIZE, h_filter_kernel,
              &h_padded_filter_kernel, FILTER_KERNEL_SIZE);
  int mem_size = sizeof(Complex) * new_size;

  // Allocate device memory for signal
  Complex *d_signal;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_signal), mem_size));
  // Copy host memory to device
  checkCudaErrors(
      hipMemcpy(d_signal, h_padded_signal, mem_size, hipMemcpyHostToDevice));

  // Allocate device memory for filter kernel
  Complex *d_filter_kernel;
  checkCudaErrors(
      hipMalloc(reinterpret_cast<void **>(&d_filter_kernel), mem_size));

  // Copy host memory to device
  checkCudaErrors(hipMemcpy(d_filter_kernel, h_padded_filter_kernel, mem_size,
                             hipMemcpyHostToDevice));

  // CUFFT plan simple API
  hipfftHandle plan;
  checkCudaErrors(hipfftPlan1d(&plan, new_size, HIPFFT_C2C, 1));

  // CUFFT plan advanced API
  hipfftHandle plan_adv;
  size_t workSize;
  long long int new_size_long = new_size;

  checkCudaErrors(hipfftCreate(&plan_adv));
  checkCudaErrors(hipfftXtMakePlanMany(plan_adv, 1, &new_size_long, NULL, 1, 1,
                                      HIP_C_32F, NULL, 1, 1, HIP_C_32F, 1,
                                      &workSize, HIP_C_32F));
  printf("Temporary buffer size %li bytes\n", workSize);

  // Transform signal and kernel
  printf("Transforming signal hipfftExecC2C\n");
  checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_signal),
                               reinterpret_cast<hipfftComplex *>(d_signal),
                               HIPFFT_FORWARD));
  checkCudaErrors(hipfftExecC2C(
      plan_adv, reinterpret_cast<hipfftComplex *>(d_filter_kernel),
      reinterpret_cast<hipfftComplex *>(d_filter_kernel), HIPFFT_FORWARD));
  
  int numBlock = 32;
  int numThreadPerBlock = 256;

  // Multiply the coefficients together and normalize the result
  printf("Launching ComplexPointwiseMulAndScale<<<%d, %d >>>\n", numBlock, numThreadPerBlock);
  ComplexPointwiseMulAndScale<<<numBlock, numThreadPerBlock>>>(d_signal, d_filter_kernel, new_size,
                                           1.0f / new_size);

  // Check if kernel execution generated and error
  getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

  // Transform signal back
  printf("Transforming signal back hipfftExecC2C\n");
  checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_signal),
                               reinterpret_cast<hipfftComplex *>(d_signal),
                               HIPFFT_BACKWARD));

  // Copy device memory to host
  Complex *h_convolved_signal = h_padded_signal;
  checkCudaErrors(hipMemcpy(h_convolved_signal, d_signal, mem_size,
                             hipMemcpyDeviceToHost));

  // Allocate host memory for the convolution result
  Complex *h_convolved_signal_ref =
      reinterpret_cast<Complex *>(malloc(sizeof(Complex) * SIGNAL_SIZE));

  // Convolve on the host
  Convolve(h_signal, SIGNAL_SIZE, h_filter_kernel, FILTER_KERNEL_SIZE,
           h_convolved_signal_ref);

  // check result
  bool bTestResult = sdkCompareL2fe(
      reinterpret_cast<float *>(h_convolved_signal_ref),
      reinterpret_cast<float *>(h_convolved_signal), 2 * SIGNAL_SIZE, 1e-5f);

  // Destroy CUFFT context
  checkCudaErrors(hipfftDestroy(plan));
  checkCudaErrors(hipfftDestroy(plan_adv));

  // cleanup memory
  free(h_signal);
  free(h_filter_kernel);
  free(h_padded_signal);
  free(h_padded_filter_kernel);
  free(h_convolved_signal_ref);
  checkCudaErrors(hipFree(d_signal));
  checkCudaErrors(hipFree(d_filter_kernel));

  exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}


/*
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv,
                   int block_size, const dim3 &dimsA,
                   const dim3 &dimsB) {
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = reinterpret_cast<float *>(malloc(mem_size_A));
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = reinterpret_cast<float *>(malloc(mem_size_B));

    // Initialize host memory
    const float valB = 0.01f;
    ConstantInit(h_A, size_A, 1.0f);
    ConstantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = reinterpret_cast<float *>(malloc(mem_size_C));

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    if (block_size == 16) {
        MatrixMulCUDA<16> <<< grid, threads >>>(d_C, d_A, d_B,
                                                dimsA.x, dimsB.x);
    } else {
        MatrixMulCUDA<32> <<< grid, threads >>>(d_C, d_A, d_B,
                                                dimsA.x, dimsB.x);
    }

    printf("done\n");

    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    checkCudaErrors(hipEventCreate(&start));

    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&stop));

    // Record the start event
    checkCudaErrors(hipEventRecord(start, NULL));

    // Execute the kernel
    int nIter = 300;

    for (int j = 0; j < nIter; j++) {
        if (block_size == 16) {
            MatrixMulCUDA<16> <<< grid, threads >>>(d_C, d_A, d_B,
                                                    dimsA.x, dimsB.x);
        } else {
            MatrixMulCUDA<32> <<< grid, threads >>>(d_C, d_A, d_B,
                                                    dimsA.x, dimsB.x);
        }
    }

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, NULL));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
                               static_cast<double>(dimsA.y) *
                               static_cast<double>(dimsB.x);
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) /
                       (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops," \
        " WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6;  // machine zero

    for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;

        if (rel_err > eps) {
