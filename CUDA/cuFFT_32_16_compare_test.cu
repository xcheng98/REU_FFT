#include "hip/hip_runtime.h"
/* 
 * A program cuFFT testing
 * Test the speed and accuracy of FP16 and FP32 calculation
 */

// System includes
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>

// Helper
/* To process command line input */
#include "nvidia_helper/helper_string.h"
/* To check cuda state */
#include "nvidia_helper/checkCudaErrors.h"

#define N 128
typedef half2 Chalf;
typedef float2 Csingle;

int DISPLAY_DATA = 1;

int run_test_FP32(int input_size){
    printf("[cuFFT32] is starting...\n");
    int mem_size = input_size*sizeof(Csingle);
    Csingle *h_idata = (Csingle *)malloc(mem_size);
    
    // Intialize the memory for the input data
    for (unsigned int i = 0; i < input_size; i++) {
        h_idata[i].x = rand() / (0.5 * static_cast<float>(RAND_MAX)) - 1;
        h_idata[i].y = rand() / (0.5 * static_cast<float>(RAND_MAX)) - 1;
    }
    if (input_size == 4) {
        h_idata[0].x = 1; h_idata[0].y = 2; h_idata[1].x = 0; h_idata[1].y = 0; 
        h_idata[2].x = 0; h_idata[2].y = 1; h_idata[3].x = -1; h_idata[3].y = 0;
    }
    if (DISPLAY_DATA == 1) {
        printf("Input data: \n");
        for (unsigned int i = 0; i < input_size; i++) {
            printf("x[%d]=(%.2f, %.2f); \n", i, h_idata[i].x, h_idata[i].y);
        }
        printf("\n"); 
    }

    // Allocate device momory for input and output
    Csingle *d_idata, *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // Copy host data to device
    checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

    // cuFFT plan
    hipfftResult result;
    hipfftHandle plan;
    size_t workSize;
    long long int input_size_long = input_size;
    result = hipfftCreate(&plan);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftCreate (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    result = hipfftXtMakePlanMany(plan, 1, &input_size_long, NULL, 1, 1, \
                         HIP_C_32F, NULL, 1, 1, HIP_C_32F, 1, \
                         &workSize, HIP_C_32F);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftXtMakePlanMany (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    printf("Temporary buffer size %li bytes\n", workSize);

    // cuFFT warm-up execution
    result = hipfftXtExec(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                          reinterpret_cast<hipfftComplex *>(d_odata), \
                          HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftExecC2C (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Measure execution time
    hipDeviceSynchronize();
    // Allocate CUDA events
    hipEvent_t start;
    checkCudaErrors(hipEventCreate(&start));
    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&stop));
    // Record the start event
    checkCudaErrors(hipEventRecord(start, NULL));
    // Repeatedly execute cuFFT
    int nIter = 300;
    for (int i = 0; i < nIter; i++){
        result = hipfftXtExec(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                              reinterpret_cast<hipfftComplex *>(d_odata), \
                              HIPFFT_FORWARD);
    }
    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, NULL));
    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));
    // Calculate performance
    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
    float msecPerFFT = msecTotal / nIter;

    // Copy Device memory to host
    Csingle *h_odata = (Csingle *)malloc(mem_size);
    checkCudaErrors(hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost));

    // Print result
    if (DISPLAY_DATA == 1) {
        printf("FFT result: \n");
        for (unsigned int i = 0; i < input_size; i++) {
            printf("x[%d]=(%.2f, %.2f); \n", i, h_odata[i].x, h_odata[i].y);
        }
        printf("\n");
    }

    // Print the performance
    printf("Performance of cuFFT32: Problem size= %d, Time= %.5f msec\n", \
        input_size,
        msecPerFFT);

    // Clean up content and memory
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    hipfftDestroy(plan);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));
    free(h_idata);
    free(h_odata);

    return 0;
}

int run_test_FP16(int input_size){
    printf("[cuFFT16] is starting...\n");
    
    // Initialize the memory for the input data
    int mem_size = input_size*sizeof(Chalf);
    Chalf *h_idata = (Chalf *)malloc(mem_size);
    for (unsigned int i = 0; i < input_size; i++) {
        h_idata[i].x = rand() / (0.5 * static_cast<float>(RAND_MAX)) - 1;
        h_idata[i].y = rand() / (0.5 * static_cast<float>(RAND_MAX)) - 1;
    }
    if (input_size == 4) {
        h_idata[0].x = 1.0f; h_idata[0].y = 2.0f;
        h_idata[1].x = 0.0f; h_idata[1].y = 0.0f; 
        h_idata[2].x = 0.0f; h_idata[2].y = 1.0f;
        h_idata[3].x = -1.0f; h_idata[3].y = 0.0f;
    }
    if (DISPLAY_DATA == 1) {
        printf("Input data: \n");
        for (unsigned int i = 0; i < input_size; i++) {
            printf("x[%d]=(%.2f, %.2f); \n", i, (float)h_idata[i].x, (float)h_idata[i].y);
        }
        printf("\n"); 
    }

    // Allocate device momory for input and output
    Chalf *d_idata, *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // Copy host data to device
    checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

    // cuFFT plan
    hipfftResult result;
    hipfftHandle plan;
    size_t workSize;
    long long int input_size_long = input_size;
    result = hipfftCreate(&plan);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftCreate (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    result = hipfftXtMakePlanMany(plan, 1, &input_size_long, NULL, 1, 1, \
                         HIP_C_16F, NULL, 1, 1, HIP_C_16F, 1, \
                         &workSize, HIP_C_16F);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftXtMakePlanMany (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    printf("Temporary buffer size %li bytes\n", workSize);

    // cuFFT warm-up execution
    result = hipfftXtExec(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                          reinterpret_cast<hipfftComplex *>(d_odata), \
                          HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftExecC2C (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Measure execution time
    hipDeviceSynchronize();
    // Allocate CUDA events
    hipEvent_t start;
    checkCudaErrors(hipEventCreate(&start));
    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&stop));
    // Record the start event
    checkCudaErrors(hipEventRecord(start, NULL));
    // Repeatedly execute cuFFT
    int nIter = 300;
    for (int i = 0; i < nIter; i++){
        result = hipfftXtExec(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                              reinterpret_cast<hipfftComplex *>(d_odata), \
                              HIPFFT_FORWARD);
    }
    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, NULL));
    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));
    // Calculate performance
    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
    float msecPerFFT = msecTotal / nIter;

    // Copy Device memory to host
    Chalf *h_odata = (Chalf *)malloc(mem_size);
    checkCudaErrors(hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost));

    // Print result
    if (DISPLAY_DATA == 1) {
        printf("FFT result: \n");
        for (unsigned int i = 0; i < input_size; i++) {
            printf("x[%d]=(%.2f, %.2f); \n", i, (float)h_odata[i].x, (float)h_odata[i].y);
        }
        printf("\n");
    }
    // Print the performance
    printf("Performance of cuFFT16: Problem size= %d, Time= %.5f msec\n", \
        input_size,
        msecPerFFT);

    // Clean up content and memory
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    hipfftDestroy(plan);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));
    free(h_idata);
    free(h_odata);
    return 0;
}

int main(int argc, char **argv)
{
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
            checkCmdLineFlag(argc, (const char **)argv, "?") ||
            checkCmdLineFlag(argc, (const char **)argv, "h")) {
        printf("Usage: -n=size (Input vector size)"
	       " -device=ID (ID > 0 for deviceID)"
               " -display=show_result (0 or 1) \n"); 
        exit(EXIT_SUCCESS);
    }
    
    // Set block size
    int block_size = 32;
 
    // Device ID by defualt is 0
    int devID = 0;
    if (checkCmdLineFlag(argc, (const char **)argv, "device")) {
        devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(devID);
    }
    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);
    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    error = hipGetDeviceProperties(&deviceProp, devID);
    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }
    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    // Input size by defualt is 8
    int n = block_size * 8;
    if (checkCmdLineFlag(argc, (const char **)argv, "n")) {
        n = getCmdLineArgumentInt(argc, (const char **)argv, "n");
    }
     
    // Set display mode
    if (checkCmdLineFlag(argc, (const char **)argv, "display")) {
        int entered_mode = getCmdLineArgumentInt(argc, (const char **)argv, "display");
        if (entered_mode == 0)  DISPLAY_DATA = 0;
    }

    printf("Problem size = %d\n", n);

    printf("[Testing of cuFFT FP32 and FP16] - Starting...\n");
    
    int test32 = run_test_FP32(n);
    int test16 = run_test_FP16(n);

    exit(test32 || test16);
} 
