#include "hip/hip_runtime.h"
/* 
 * A program cuFFT testing
 * Test the speed and accuracy of FP16 and FP32 calculation
 */

// System includes
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>

// Helper
/* To process command line input */
# include "nvidia_helper/helper_string.h"

#define N 128
typedef half2 Chalf;
typedef float2 Csingle;

int run_test_FP32(int input_size){
    printf("[cuFFT32] is starting...\n");
    int mem_size = input_size*sizeof(Csingle);
    Csingle *h_idata = (Csingle *)malloc(mem_size);
    
    // Intialize the memory for the input data
    for (unsigned int i = 0; i < input_size; i++) {
        h_idata[i].x = rand() / (0.5 * static_cast<float>(RAND_MAX)) - 1;
        h_idata[i].y = rand() / (0.5 * static_cast<float>(RAND_MAX)) - 1;
    }
    h_idata[0].x = 1; h_idata[0].y = 2; h_idata[1].x = 0; h_idata[1].y = 0; 
    h_idata[2].x = 0; h_idata[2].y = 1; h_idata[3].x = -1; h_idata[3].y = 0;
    for (unsigned int i = 0; i < input_size; i++) {
        printf("x[%d]=(%f, %f); ", i, h_idata[i].x, h_idata[i].y);
    }
    printf("\n"); 

    // Allocate device momory for input and output
    Csingle *d_idata, *d_odata;
    hipError_t error;
    error = hipMalloc((void **) &d_idata, mem_size);
    if (error != hipSuccess)
    {
        printf("hipMalloc d_idata returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **) &d_odata, mem_size);
    if (error != hipSuccess)
    {
        printf("hipMalloc d_odata returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Copy host data to device
    error = hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_idata,h_idata) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // cuFFT plan
    hipfftResult result;
    hipfftHandle plan;
    size_t workSize;
    long long int input_size_long = input_size;
    result = hipfftCreate(&plan);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftCreate (plan) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    result = hipfftXtMakePlanMany(plan, 1, &input_size_long, NULL, 1, 1, \
                         HIP_C_32F, NULL, 1, 1, HIP_C_32F, 1, \
                         &workSize, HIP_C_32F);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftXtMakePlanMany (plan) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    printf("Temporary buffer size %li bytes\n", workSize);

    // cuFFT warm-up execution
    result = hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                          reinterpret_cast<hipfftComplex *>(d_odata), \
                          HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftExecC2C (plan) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Measure execution time
    hipDeviceSynchronize();
    hipEvent_t start;
    checkCudaErrors(hipEventCreate(&start));
    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&stop));


    // Copy Device memory to host
    Csingle *h_odata = (Csingle *)malloc(mem_size);
    error = hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_odata,d_odata) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Print result
    for (unsigned int i = 0; i < input_size; i++) {
        printf("x[%d]=(%f, %f); ", i, h_odata[i].x, h_odata[i].y);
    }
    printf("\n"); 

    // Clean up content and memory
    hipfftDestroy(plan);
    hipFree(d_idata);
    hipFree(d_odata);
    free(h_idata);
    free(h_odata);

    return 0;
}

int run_test_FP16(int input_size){
    printf("[cuFFT16] is starting...\n");
    Chalf *h_idata = (Chalf *)malloc(input_size*sizeof(Chalf));
    Chalf *d_idata, *d_odata;
    
    printf("size: %d\n", sizeof(Chalf));    
    return 0;
}

int main(int argc, char **argv)
{
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
            checkCmdLineFlag(argc, (const char **)argv, "?")) {
        printf("Usage: -n=size (Input vector size) -device=ID (ID > 0 for deviceID)\n"); 
        exit(EXIT_SUCCESS);
    }
    
    // Set block size
    int block_size = 32;
 
    // Device ID by defualt is 0
    int devID = 0;
    if (checkCmdLineFlag(argc, (const char **)argv, "device")) {
        devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(devID);
    }
    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);
    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    error = hipGetDeviceProperties(&deviceProp, devID);
    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }
    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    // Input size by defualt is 8
    int n = block_size * 8;
    if (checkCmdLineFlag(argc, (const char **)argv, "n")) {
        n = getCmdLineArgumentInt(argc, (const char **)argv, "n");
    }
    printf("Size = %d\n", n);
     

    printf("[Testing of cuFFT FP32 and FP16] - Starting...\n");
    
    int test32 = run_test_FP32(n);
    int test16 = run_test_FP16(n);

    exit(test32 || test16);
} 
