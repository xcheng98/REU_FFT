#include "hip/hip_runtime.h"
/*
 * Implementing the FFT algorithm for general input
 * Input should be fp32 vectors with size equals to the power of 4
 * Number of vectors is given by BATCH (B)
 * Recursive algorithm, base case is fft4
 * Combine all components in one file
 * Version after multiple optimizations
 * This implementation is without matrix and vector
 */

#include "my_include_combined.h"


#define PI 3.14159265
#define EPS 0.0000001192f


const float UPPER_BOUND = 1.0f;
const int BATCH = 16;
const int SIZE = 256;


FFT_S gfft(int N, float* X_re, float* X_im, float* FX_re, float* FX_im, int B);
 
__global__ void myTranspose(int m, int n, float* input, float* output, int B);

__global__ void multiply_twiddle(int N, int m, int n, float* matrix_re, float* matrix_im, int B);

FFT_S init_F4();

FFT_S fft4(float* X_re, float* X_im, float* FX_re, float* FX_im, int B);

__global__ void mySplit(int N, float* X, half* Xhi, half* Xlo, float* s1, float* s2, int B, float* Xtemp);

__global__ void myAccumulate(int N, float* X1, float* X2, float* alpha, float* R1, float* R2, int B);

FFT_S fft4_transposed(int M, float* X_re, float* X_im, float* FX_re, float* FX_im, int B);

__global__ void mySplit_transposed(int n, int M, float* X, half* Xhi, half* Xlo, float* s1, float* s2, int B, float* Xtemp);

__global__ void myAccumulate_transposed(int n, int M, float* X1, float* X2, float* alpha, float* R1, float* R2, int B);


half* F4_re;
half* F4_im;
float* buffer;


int main()
{
    int mem_size;
    FFT_S status;

    // Allocate unified memory for input and output matrix
    float* input_re, input_im, output_re, output_im;
    mem_size = BATCH * SIZE * sizeof(float);
    checkCudaErrors(hipMallocManaged((void **) &input_re, mem_size));
    checkCudaErrors(hipMallocManaged((void **) &input_im, mem_size));
    checkCudaErrors(hipMallocManaged((void **) &output_re, mem_size));
    checkCudaErrors(hipMallocManaged((void **) &output_im, mem_size));

    // Initialize the input data
    srand(time(NULL));
    printf("The input is: \n");
    for (int j = 0; j < BATCH; j++){
        printf("Vector %d: \n", j);
        for (int i = 0; i < SIZE; i++){
            input_re[i + j * SIZE] = (float)rand() / (float)(RAND_MAX) * 2 * UPPER_BOUND - UPPER_BOUND;
            input_im[i + j * SIZE] = (float)rand() / (float)(RAND_MAX) * 2 * UPPER_BOUND - UPPER_BOUND;
            input_re[i + j * SIZE] = (float)i;
            input_im[i + j * SIZE] = 0.0f;
            printf("X[%d] = (%.10f, %.10f) \n", i, input_re[i + j * SIZE], input_re[i + j * SIZE]);
        }
        printf("\n");
    }
    
    // Allocate unified memory for the buffer (array of float)
    mem_size = SIZE * BATCH * sizeof(float);
    checkCudaErrors(hipMallocManaged((void **) &buffer, mem_size));

    // Initialize Fourier matrix
    status = init_F4();
    if (status != FFT_SUCCESS){
        fprintf(stderr, "!!!!! Matrix initialization error (Fourier matrix).\n");
        exit(1);
    }

    // Call gfft function
    status = gfft(SIZE, input_re, input_im, output_re, output_im, BATCH);
    if (status != FFT_SUCCESS){
        fprintf(stderr, "!!!!! gFFT execution error.\n");
        exit(1);
    }

    // Print result
    printf("Result: \n");
    for (int j = 0; j < BATCH; j++){
        printf("Resulting vector %d: \n", j);
        for (int i = 0; i < SIZE; i++){
            printf("FX[%d] = (%.10f, %.10f) \n", i, output_re[i + j * SIZE], output_im[i + j * SIZE]);
        }
    }

    // Deallocate unified memory
    checkCudaErrors(hipFree(input_re));
    checkCudaErrors(hipFree(input_im));
    checkCudaErrors(hipFree(output_re));
    checkCudaErrors(hipFree(output_im));

    exit(0);
}


FFT_S gfft(int N, float* X_re, float* X_im, float* FX_re, float* FX_im, int B) 
{
    // Base case
    if (N == 4) {
        return fft4(X_re, X_im, FX_re, FX_im, B);
    }

    // Status variable declaration
    hipblasStatus_t status;
    hipblasHandle_t handle;
    FFT_S fft_status;
    hipError_t cerror;

    // Declare temp variable for buffer swapping
    float* temp;

    // Initialize cublas
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!!! CUBLAS initialization error.\n");
        return FFT_FAILURE;
    }

    // Transpose input matrix: 4 * (N/4*B) --> (N/4) * (4*B)
    // First store the result in buffer to avoid racing condition
    //// Set grid and block size
    dim3 threadsPerBlock1(4, 16);
    dim3 blockPerGrid1(B, (N / 4 + 15)/16); // Make sure blocks are enough

    //// Transpose real matrix
    myTranspose<<<blockPerGrid1, threadsPerBlock1>>>(4, N / 4, X_re, buffer, B);
    cerror = hipGetLastError();
    if (cerror != hipSuccess)
    {
        fprintf(stderr, "!!!!! CUDA error: %s during transposition of real matrix.\n", hipGetErrorString(cerror));
        return FFT_FAILURE;
    }

    //// Swap FX_re and buffer to store the transposition result in FX_re
    temp = FX_re; FX_re = buffer; buffer = temp;

    //// Transpose imaginary matrix
    myTranspose<<<blockPerGrid1, threadsPerBlock1>>>(4, N / 4, X_im, buffer, B);
    cerror = hipGetLastError();
    if (cerror != hipSuccess)
    {
        fprintf(stderr, "!!!!! CUDA error: %s during transposition of imaginary matrix.\n", hipGetErrorString(cerror));
        return FFT_FAILURE;
    }
    ////// Swap FX_im and buffer to store the transposition result in FX_im
    temp = FX_im; FX_im = buffer; buffer = temp;

    // Wait for GPU to finish work
    hipDeviceSynchronize();

    // Recursively call gfft function, not! using buffer matrix
    fft_status = gfft(N / 4, 4 * B, FX_re, FX_im, FX_re, FX_im);
    if (fft_status != FFT_SUCCESS){
        fprintf(stderr, "!!!!! Function execution error (recursively call gfft).\n");
        return FFT_FAILURE;
    }

    // Wait for GPU to finish work
    hipDeviceSynchronize();

    // Multiplicate each element with twiddle factor
    //// Set grid and block size
    dim3 threadsPerBlock2(4, 16);
    dim3 blockPerGrid2(B, (N / 4 + 15)/16); // Make sure blocks are enough

    //// Call kernel function
    multiply_twiddle<<<blockPerGrid2, threadsPerBlock2>>>(N, N/4, 4, FX_re, FX_im, B);
    cerror = hipGetLastError();
    if (cerror != hipSuccess)
    {
        fprintf(stderr, "!!!!! CUDA error: %s during twiddle factor multiplication.\n", hipGetErrorString(cerror));
        return FFT_FAILURE;
    }

    // Call the optimized fft4 function to avoid transposition
    fft_status = fft4_transposed(N / 4, FX_re, FX_im, FX_re, FX_im, B);
    if (fft_status != FFT_SUCCESS){
        fprintf(stderr, "!!!!! Function execution error (calling fft4_transposed).\n");
        return FFT_FAILURE;
    } 

    // Shutdown cublas
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!!! shutdown error.\n");
        return FFT_FAILURE;
    }

    return FFT_SUCCESS;
}

/* 
 * Transpose every input matrix of size m * n
 * Number of matrices is given by B 
 * Every matrix in a batch is transposed independently
 * Input is expected to be matrix of size m * (n * B)
 * Output is expected to be matrix of size n * (m * B)
 * The grid size is expected to be B in horizontal dimension
 * Usage: transpose a matrix of size 4 * (N/4 * B) to (N/4) * (4 * B)
 * */
__global__ void myTranspose(int m, int n, float* input, float* output, int B)
{
    // Calculate position in the OUTPUT matrix (0 based)
    int j = threadIdx.x; // Column number within a matrix, expected to be 0, 1, 2, 3
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Row number
    int matrix_id = blockIdx.x; // The index of matrix in the batch

    if (i < n && j < m && matrix_id < B){
        output[matrix_id * m * n + j * n + i] = input[matrix_id * m * n + i * m + j];
    }
}

/* 
 * Multifly every element of the input matrix with the twiddle factor
 * Every matrix in a batch is processed independently
 * Block and thread layout should be 2D, and the total dimension is expected to be (m, n * B)
 * n is expected to be 4
 * result.re(i, j) [0 based] = xre(i, j) * cos(2pi/N * i * j) + xim(i, j) * sin(2pi/N * i * j)
 * result.im(i, j) [0 based] = -xre(i, j) * sin(2pi/N * i * j) + xim(i, j) * cos(2pi/N * i * j)
 * ONLY that thread will access the particular matrix_re and matrix_im, so buffer is not needed
 * */
__global__ void multiply_twiddle(int N, int m, int n, float* matrix_re, float* matrix_im, int B)
{
    // Calculate position
    int j = threadIdx.x; // Column number within a matrix, 0 to 3 in radix 4
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Row number within a matrix
    int matrix_id = blockIdx.x; // Index of matrix in the batch

    if (i < m && j < n && matrix_id < B){
        // Per-thread local variables
        int index = matrix_id * N + j * m + i;
        float tw_re = cos(2 * PI / N * i * j);
        float tw_im = sin(2 * PI / N * i * j);
        float result_re = matrix_re[index] * tw_re + matrix_im[index] * tw_im;
        float result_im = -1.0f * matrix_re[index] * tw_im + matrix_im[index] * tw_re;

        matrix_re[index] = result_re;
        matrix_im[index] = result_im;
    }
}


/* 
 * Perform fft on every length-4 vector
 * Batch size is given by B
 * Internally split every FP32 input into two FP16 vectors
 * Combine them together after FFT
 * */
FFT_S fft4(float* X_re, float* X_im, float* FX_re, float* FX_im, int B) 
{
    // Variable declaration
    hipblasStatus_t status;
    hipblasHandle_t handle;
    hipError_t cerror;
    float alpha = 1.0f, beta = 0.0f; 
    float* scales; // = re_s1, re_s2, im_s1, im_s2;
    half* X_split; // = X_re_hi, X_re_lo, X_im_hi, X_im_lo;
    float *result1, *result2; // F4_re * X_split, F4_im * X_split

    // Initialize cublas
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!!! CUBLAS initialization error\n");
        return FFT_FAILURE;
    }
    status = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH); // allow Tensor Core
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!!! CUBLAS setting math mode error\n");
        return FFT_FAILURE;
    }

    //  Allocate unified memory
    checkCudaErrors(hipMallocManaged((void **) &scales, B * 4 * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **) &X_split, 4 * B * 4 * sizeof(half)));
    checkCudaErrors(hipMallocManaged((void **) &result1, 4 * B * 4 * sizeof(result1[0])));
    checkCudaErrors(hipMallocManaged((void **) &result2, 4 * B * 4 * sizeof(result2[0])));

    // Split input
    //// Define segmentation pointers for convenience
    half* X_re_hi = X_split + 4 * B * 0;
    half* X_re_lo = X_split + 4 * B * 1;
    half* X_im_hi = X_split + 4 * B * 2;
    half* X_im_lo = X_split + 4 * B * 3;
    float* re_s1 = scales + B * 0;
    float* re_s2 = scales + B * 1;
    float* im_s1 = scales + B * 2;
    float* im_s2 = scales + B * 3;

    //// Call the splitting kernel
    int numThreads = 64;
    int numBlocks = (B + 63) / 64;
    mySplit<<<numBlocks, numThreads>>>(4, X_re, X_re_hi, X_re_lo, re_s1, re_s2, B, buffer);
    mySplit<<<numBlocks, numThreads>>>(4, X_im, X_im_hi, X_im_lo, im_s1, im_s2, B, buffer);
    cerror = hipGetLastError();
    if (cerror != hipSuccess)
    {
        fprintf(stderr, "!!!!! CUDA error: %s during fft4 splitting\n", hipGetErrorString(cerror));
        return FFT_FAILURE;
    }
  
    // Matrix multiplication with Fourier matrix
    //// Call cublas gemm on F4_re
    status = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, B * 4, 4, &alpha, 
        F4_re,HIP_R_16F, 4, X_split, HIP_R_16F, 4, &beta,
        result1, HIP_R_32F, 4, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!!! CUBLAS kernel execution error (F4_re * X_split).\n");
        return FFT_FAILURE;
    }

    //// Call cublas gemm on F4_im
    status = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, B * 4, 4, &alpha,
        F4_im, HIP_R_16F, 4, X_split, HIP_R_16F, 4, &beta,
        result2, HIP_R_32F, 4, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!!! CUBLAS kernel execution error (F4_im * X_split).\n");
        return FFT_FAILURE;
    }

    // Rescale the result and combine them together
    //// Set grid and block size
    dim3 threadsPerBlock(16, 4);
    dim3 blocksPerGrid((B+15)/16, 1);

    //// call kernel function (FX_re and FX_im will be zero-initialized)
    myAccumulate<<<blocksPerGrid, threadsPerBlock>>>(4, result1, result2, scales, FX_re, FX_im, B);
    cerror = hipGetLastError();
    if (cerror != hipSuccess)
    {
        fprintf(stderr, "!!!!! CUDA error: %s during fft4 accumulation\n", hipGetErrorString(cerror));
        return FFT_FAILURE;
    }

    // Deallocate unified memory
    checkCudaErrors(hipFree(scales));
    checkCudaErrors(hipFree(X_split));
    checkCudaErrors(hipFree(result1));
    checkCudaErrors(hipFree(result2));

    // Shutdown cublas
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!!! shutdown error\n");
        return FFT_FAILURE;
    }

    return FFT_SUCCESS;
}

/* 
 * Initialize Fourier matrix
 * Allocate unified memory and set value for F4_re and F4_im
 * */
FFT_S init_F4()
{
    // Allocate unified memory for Fourier Matrix
    int mem_size = 16 * sizeof(half);
    checkCudaErrors(hipMallocManaged((void **) &F4_re, mem_size));
    checkCudaErrors(hipMallocManaged((void **) &F4_im, mem_size));

    F4_re.element(1, 1) = 1.0f;
    F4_re.element(2, 1) = 1.0f;
    F4_re.element(3, 1) = 1.0f;
    F4_re.element(4, 1) = 1.0f;
    F4_re.element(1, 2) = 1.0f;
    F4_re.element(2, 2) = 0.0f;
    F4_re.element(3, 2) =-1.0f;
    F4_re.element(4, 2) = 0.0f;
    F4_re.element(1, 3) = 1.0f;
    F4_re.element(2, 3) =-1.0f;
    F4_re.element(3, 3) = 1.0f;
    F4_re.element(4, 3) =-1.0f;
    F4_re.element(1, 4) = 1.0f;
    F4_re.element(2, 4) = 0.0f;
    F4_re.element(3, 4) =-1.0f;
    F4_re.element(4, 4) = 0.0f;

    F4_im.element(1, 1) = 0.0f;
    F4_im.element(2, 1) = 0.0f;
    F4_im.element(3, 1) = 0.0f;
    F4_im.element(4, 1) = 0.0f;
    F4_im.element(1, 2) = 0.0f;
    F4_im.element(2, 2) =-1.0f;
    F4_im.element(3, 2) = 0.0f;
    F4_im.element(4, 2) = 1.0f;
    F4_im.element(1, 3) = 0.0f;
    F4_im.element(2, 3) = 0.0f;
    F4_im.element(3, 3) = 0.0f;
    F4_im.element(4, 3) = 0.0f;
    F4_im.element(1, 4) = 0.0f;
    F4_im.element(2, 4) = 1.0f;
    F4_im.element(3, 4) = 0.0f;
    F4_im.element(4, 4) =-1.0f;

    return FFT_SUCCESS;
}

/* 
 * float* X (N*B), half* Xhi (N*B), half* Xlo (N*B)
 * float* s1, float* s2
 * int N, int B. N is always 4
 * Grid and dim size should be 1D, total size = B
 * All data should be in unified memory or device memory
 * */
__global__ void mySplit(int N, float* X, half* Xhi, half* Xlo, float* s1, float* s2, int B, float* Xtemp)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < B){
        // Calculate scaling factor 1
        float scale1 = 0.0f;
        for (int i = 0; i < N; i++){
            float norm = (float) fabs(X[i + idx * N]);
            if (norm > scale1) scale1 = norm;
        }
        
        // If all number are zero, skip
        if (scale1 == 0.0f){
            s1[idx] = 0.0f;
            s2[idx] = 0.0f;
            for (int i = 0; i < N; i++){
                Xhi[i + idx * N] = Xlo[i + idx * N] = 0.0f;
            }
        }
        else
        {
            // Restrict scale range
            if (scale1 < EPS) scale1 = EPS;
            if (scale1 > 1.0f/EPS) scale1 = 1.0f/EPS;
            s1[idx] = scale1;

            // Scale the high half
            for (int i = 0; i < N; i++){
                Xtemp[i + idx * N] = X[i + idx * N]/scale1;
                Xhi[i + idx * N] = (half)(Xtemp[i + idx * N]);
                // Use Xtemp to store the residual
                Xtemp[i + idx * N] = X[i + idx * N] - scale1 * (float)(Xhi[i + idx * N]);
            }

           // Calculate the lower scaling factor
            float scale2 = 0.0f;
            for (int i = 0; i < N; i++){
                float norm = (float) fabs(Xtemp[i + idx * N]);
                if (norm > scale2) scale2 = norm;
            }
        
            // If all number are zero, skip
            if (scale2 == 0.0f){
                s2[idx] = 0.0f;
                for (int i = 0; i < N; i++){
                    Xlo[i + idx * N] = 0.0f;
                }
            }
            else
            {
                // Restrict scale range
                if (scale2 < EPS) scale2 = EPS;
                if (scale2 > 1.0f/EPS) scale2 = 1.0f/EPS;
                s2[idx] = scale2;

                for (int i = 0; i < N; i++){
                Xlo[i + idx * N] = (half) (Xtemp[i + idx * N] / scale2);
                }
            }
        }
    }
}

/* 
 * N is number of elements in one column (expected to be 4)
 * X1, X2 are 4 * (B * 4) column-major matrix. Inner order is by batch. Outer order is Re_hi, Re_lo, Im_hi, Im_lo
 * alpha is B * 4 array. Inner order is by batch. Outer order is re_s1, re_s2, im_s1, im_s2
 * R1, R2 are 4 * B matrix
 * B is batch size
 * */
__global__ void myAccumulate(int N, float* X1, float* X2, float* alpha, float* R1, float* R2, int B)
{

    int i = blockIdx.y * blockDim.y + threadIdx.y; // row number
    int j = blockIdx.x * blockDim.x + threadIdx.x; // column number

    if (i < N && j < B){
        R1[i + j * N] = R2[i + j * N] = 0.0f;
        R1[i + j * N] += alpha[j] * X1[i + j * N];
        R1[i + j * N] += alpha[j + B] * X1[i + j * N + N * B];
        R1[i + j * N] += -1.0f * alpha[j + 2*B] * X2[i + j * N + N * 2 * B];
        R1[i + j * N] += -1.0f * alpha[j + 3*B] * X2[i + j * N + N * 3 * B];
        R2[i + j * N] += alpha[j] * X2[i + j * N];
        R2[i + j * N] += alpha[j + B] * X2[i + j * N + N * B];
        R2[i + j * N] += alpha[j + 2*B] * X1[i + j * N + N * 2 * B];
        R2[i + j * N] += alpha[j + 3*B] * X1[i + j * N + N * 3 * B];
    }
}



/* 
 * Perform fft4 assuming the input is in the transposed layout
 * M is the number of rows
 * 4 * B is the number of columns
 * Note that the fourier matrix is symmetric
 */
FFT_S fft4_transposed(int M, float* X_re, float* X_im, float* FX_re, float* FX_im, int B) 
{


    // Variable declaration
    hipblasStatus_t status;
    hipblasHandle_t handle;
    hipError_t cerror;

    //// Unified variables
    float *scales; // = *re_s1, *re_s2, *im_s1, *im_s2;
    half *X_split; // = *X_re_hi, *X_re_lo, *X_im_hi, *X_im_lo;
    float *result1, *result2; // Store the intermediate result
    //// Scaling variables
    float alpha = 1.0f, beta = 0.0f; 

    // Initialize cublas
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return FFT_FAILURE;
    }

    status = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH); // allow Tensor Core
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS setting math mode error\n");
        return FFT_FAILURE;
    }

    //  Allocate unified memory with 0 initialization
    checkCudaErrors(hipMallocManaged((void **) &scales, M * B * 4 * sizeof(float)));
    checkCudaErrors(hipMemset(scales, 0.0f, M * B * 4 * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **) &X_split, M * 4 * B * 4 * sizeof(half)));
    checkCudaErrors(hipMemset(X_split, 0.0f, M * 4 * B * 4 * sizeof(half)));
    checkCudaErrors(hipMallocManaged((void **) &result1, M * 4 * B * 4 * sizeof(result1[0])));
    checkCudaErrors(hipMemset(result1, 0.0f, M * 4 * B * 4 * sizeof(result1[0])));
    checkCudaErrors(hipMallocManaged((void **) &result2, M * 4 * B * 4 * sizeof(result2[0])));
    checkCudaErrors(hipMemset(result2, 0.0f, M * 4 * B * 4 * sizeof(result2[0])));

    // Split input
    //// Initialize Matrix and Vector data structure to store split result
    half* X_re_hi;
    X_re_hi.width = 4 * B;
    X_re_hi.height = M;
    X_re_hi = X_split + M * 4 * B * 0;

    half* X_re_lo;
    X_re_lo.width = 4 * B;
    X_re_lo.height = M;
    X_re_lo = X_split + M * 4 * B * 1;

    half* X_im_hi;
    X_im_hi.width = 4 * B;
    X_im_hi.height = M;
    X_im_hi = X_split + M * 4 * B * 2;

    half* X_im_lo;
    X_im_lo.width = 4 * B;
    X_im_lo.height = M;
    X_im_lo = X_split + M * 4 * B * 3;

    float* re_s1;
    re_s1.size = M * B;
    re_s1 = scales + M * B * 0;

    float* re_s2;
    re_s2.size = M * B;
    re_s2 = scales + M * B * 1;

    float* im_s1;
    im_s1.size = M * B;
    im_s1 = scales + M * B * 2;

    float* im_s2;
    im_s2.size = M * B;
    im_s2 = scales + M * B * 3;

    //// Call splitting function
    dim3 threadsPerBlock1(4, 16);
    dim3 BlocksPerGrid1((B + 3)/4, (M + 15)/16);
    mySplit_transposed<<<4, M, BlocksPerGrid1, threadsPerBlock1>>>(X_re, X_re_hi, X_re_lo, re_s1, re_s2, B, buffer);
    mySplit_transposed<<<4, M, BlocksPerGrid1, threadsPerBlock1>>>(X_im, X_im_hi, X_im_lo, im_s1, im_s2, B, buffer);
    cerror = hipGetLastError();
    if (cerror != hipSuccess)
    {
        printf("CUDA error: %s during splitting in fft4_transposed\n", hipGetErrorString(cerror));
        return FFT_FAILURE;
    }
  
 
    // Call cublas function and finish Matrix multiplication calculation
    // The order of multiplicands are reversed
    //// Define batched offset
    long long int stride = M * 4;

    //// Call cublas batched gemm on F4_re

    status = hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, 4, 4, &alpha, X_split,
                        HIP_R_16F, M, stride, F4_re, HIP_R_16F, 4, 0, &beta, result1, HIP_R_32F, M, stride, B * 4, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS kernel execution error: %d in fft4_transposed ((c, d) * a).\n", status);
        printf("Error is: %d\n", status == HIPBLAS_STATUS_NOT_SUPPORTED);
        return FFT_FAILURE;
    }

    //// Call cublas gemm on F4_im
    status = hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, 4, 4, &alpha, X_split,
                        HIP_R_16F, M, stride, F4_im, HIP_R_16F, 4, 0, &beta, result2, HIP_R_32F, M, stride, B * 4, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! CUBLAS kernel execution error in fft4_transposed ((c, d) * b).\n");
        return FFT_FAILURE;
    }


    // Scale, combine and get result, add to output
    //// Set grid and block size
    dim3 threadsPerBlock2(16, 16);
    dim3 BlocksPerGrid2((4 * B + 15)/16, (M + 15)/16);

    //// call kernel function (buffer is zero-initialized inside)
    myAccumulate_transposed<<<BlocksPerGrid2, threadsPerBlock2>>>(4, M, result1, result2, scales, FX_re, FX_im, B);
    cerror = hipGetLastError();
    if (cerror != hipSuccess)
    {
        printf("CUDA error: %s during accumulation in fft4_transposed\n", hipGetErrorString(cerror));
        return FFT_FAILURE;
    }


    // Deallocate unified memory
    if (hipFree(scales) != hipSuccess) {
        fprintf(stderr, "!!!! unified memory free error (free scales vector)\n");
        return FFT_FAILURE;
    }

    if (hipFree(X_split) != hipSuccess) {
        fprintf(stderr, "!!!! unified memory free error (free split result matrix)\n");
        return FFT_FAILURE;
    }

    if (hipFree(result1) != hipSuccess) {
        fprintf(stderr, "!!!! unified memory free error (free result 1 Matrix)\n");
        return FFT_FAILURE;
    }

    if (hipFree(result2) != hipSuccess) {
        fprintf(stderr, "!!!! unified memory free error (free result 2 Matrix)\n");
        return FFT_FAILURE;
    }

    // Shutdown cublas
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error (A)\n");
        return FFT_FAILURE;
    }

    return FFT_SUCCESS;
}

/* 
 * float* X (M * (n * B)), half* Xhi (M * (n * B)), half* Xlo (M * (n * B))
 * float* s1 of size M * B, float* s2 of size M * B
 * int n, int M, int B. n is expected to be 4, M = N / 4
 * Grid and dim size should be 2D, total size = M * B
 * All data should be in unified memory or device memory
 * */
__global__ void mySplit_transposed(int n, int M, float* X, half* Xhi, half* Xlo, float* s1, float* s2, int B, float* Xtemp)
{

    int rowIdx = blockIdx.y * blockDim.y + threadIdx.y; // Row number (max M)
    int blockNum = blockIdx.x * blockDim.x + threadIdx.x; // 'Column number' (max B)

    if (rowIdx < M && blockNum < B){
        /* Data to be manipulated:
         *  X, Xhi, Xlo (rowIdx, blockIdx * n +0+1+2+3) = X, Xhi, Xlo[rowIdx + blockIdx * n * M + 0/1/2/3 * M]
         * s1, s2 (rowIdx, blockIdx) = s1, s2[rowIdx + blockIdx * M]
         */
        int offset = rowIdx + blockNum * n * M;
        int stride = M;
        int factor_idx = rowIdx + blockNum * M;

        // Calculate scaling factor 1
        float scale1 = 0.0f;
        for (int i = 0; i < n; i++){
            float norm = (float) fabs(X[offset + i * stride]);
            if (norm > scale1) scale1 = norm;
        }
        
        // If all number are zero, skip
        if (scale1 == 0.0f){
            s1[factor_idx] = 0.0f;
            s2[factor_idx] = 0.0f;
            for (int i = 0; i < n; i++){
                Xhi[offset + i * stride] = Xlo[offset + i * stride] = 0.0f;
            }
        }
        else
        {
            // Restrict scale range
            if (scale1 < EPS) scale1 = EPS;
            if (scale1 > 1.0f/EPS) scale1 = 1.0f/EPS;
            s1[factor_idx] = scale1;

            // Scale the high half
            for (int i = 0; i < n; i++){
                Xtemp[offset + i * stride] = X[offset + i * stride]/scale1;
                Xhi[offset + i * stride] = (half)(Xtemp[offset + i * stride]);
                // Use Xtemp to store the residual
                Xtemp[offset + i * stride] = X[offset + i * stride] - scale1 * (float)(Xhi[offset + i * stride]);
            }

           // Calculate the lower scaling factor
            float scale2 = 0.0f;
            for (int i = 0; i < n; i++){
                float norm = (float) fabs(Xtemp[offset + i * stride]);
                if (norm > scale2) scale2 = norm;
            }
        
            // If all number are zero, skip
            if (scale2 == 0.0f){
                s2[factor_idx] = 0.0f;
                for (int i = 0; i < n; i++){
                    Xlo[offset + i * stride] = 0.0f;
                }
            }
            else
            {
                // Restrict scale range
                if (scale2 < EPS) scale2 = EPS;
                if (scale2 > 1.0f/EPS) scale2 = 1.0f/EPS;
                s2[factor_idx] = scale2;

                for (int i = 0; i < n; i++){
                Xlo[offset + i * stride] = (half) (Xtemp[offset + i * stride] / scale2);
                }
            }
        }
    }
}


/* 
 * X1, X2 are M * (4 * B * 4) matrix. The inner-most column order is by element in a unit. Then by batch. Outer order is Re_hi, Re_lo, Im_hi, Im_lo
 * alpha is a M * B * 4 array. Inner most order is by rows. Then by batch. Outer order is re_s1, re_s2, im_s1, im_s2
 * R1, R2 are M * (4 * B) matrix
 * n is number of elements in one unit (expected to be 4)
 * M is number of rows, B is batch size
 * */
__global__ void myAccumulate_transposed(int n, int M, float* X1, float* X2, float* alpha, float* R1, float* R2, int B)
{

    int i = blockIdx.y * blockDim.y + threadIdx.y; // row number
    int j = blockIdx.x * blockDim.x + threadIdx.x; // column number

    if (i < M && j < 4 * B){
        int result_idx = i + j * M;
        int e_stride = M * 4 * B; // Stride for elements, e.g. from Re_hi to Re_lo
        int factor_idx = i + j / 4 * M;
        int f_stride = M * B; // Stride for factors, e.g. from re_s1 to re_s2
        R1[result_idx] = R2[result_idx] = 0.0f;

        R1[result_idx] += alpha[factor_idx] * X1[result_idx];
        R1[result_idx] += alpha[factor_idx + f_stride] * X1[result_idx + e_stride];
        R1[result_idx] += -1.0f * alpha[factor_idx + 2*f_stride] * X2[result_idx + 2*e_stride];
        R1[result_idx] += -1.0f * alpha[factor_idx + 3*f_stride] * X2[result_idx + 3*e_stride];
        R2[result_idx] += alpha[factor_idx] * X2[result_idx];
        R2[result_idx] += alpha[factor_idx + f_stride] * X2[result_idx + e_stride];
        R2[result_idx] += alpha[factor_idx + 2*f_stride] * X1[result_idx + 2*e_stride];
        R2[result_idx] += alpha[factor_idx + 3*f_stride] * X1[result_idx + 3*e_stride];
    }
}
