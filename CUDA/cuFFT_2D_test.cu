#include "hip/hip_runtime.h"
/* 
 * A program cuFFT testing
 * Test the speed and accuracy of FP16 and FP32 calculation
 */

// System includes
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>

// Helper
/* To process command line input */
#include "nvidia_helper/helper_string.h"
/* To check cuda state */
#include "nvidia_helper/checkCudaErrors.h"

#define N 128
typedef half2 Chalf;
typedef float2 Csingle;

int DISPLAY_DATA = 1;

int run_test_1D(int input_size){
    printf("[cuFFT 1D] is starting...\n");
    int mem_size = input_size * input_size * sizeof(Csingle);
    Csingle *h_idata = (Csingle *)malloc(mem_size);
    
    // Intialize the memory for the input data
    for (unsigned int i = 0; i < input_size; i++) {    
        for (unsigned int j = 0; j < input_size; j++) {
            h_idata[i*input_size + j].x = rand() / (0.5 * static_cast<float>(RAND_MAX)) - 1;
            h_idata[i*input_size + j].y = rand() / (0.5 * static_cast<float>(RAND_MAX)) - 1;
        }
        if (input_size == 4) {
            h_idata[i*input_size + 0].x = 1; h_idata[i*input_size + 0].y = 2;
            h_idata[i*input_size + 1].x = 0; h_idata[i*input_size + 1].y = 0; 
            h_idata[i*input_size + 2].x = 0; h_idata[i*input_size + 2].y = 1;
            h_idata[i*input_size + 3].x = -1; h_idata[i*input_size + 3].y = 0;
        }
    }
    if (DISPLAY_DATA == 1) {
        printf("Input data: \n");
        for (unsigned int i = 0; i < input_size; i++) {    
            for (unsigned int j = 0; j < input_size; j++) {
                printf("x[%d, %d]=(%.2f, %.2f); \n", j, i, h_idata[i*input_size + j].x, h_idata[i*input_size + j].y);
            }
            printf("\n"); 
        }
    }
    // Allocate device momory for input and output
    Csingle *d_idata, *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // Copy host data to device
    checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

    // cuFFT plan
    hipfftResult result;
    hipfftHandle plan;
    size_t workSize;
    long long int input_size_long = input_size;
    result = hipfftCreate(&plan);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftCreate (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    result = hipfftXtMakePlanMany(plan, 1, &input_size_long, NULL, 1, 1, \
                         HIP_C_32F, NULL, 1, 1, HIP_C_32F, input_size, \
                         &workSize, HIP_C_32F);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftXtMakePlanMany (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    printf("Temporary buffer size %li bytes\n", workSize);

    // cuFFT warm-up execution
    result = hipfftXtExec(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                          reinterpret_cast<hipfftComplex *>(d_odata), \
                          HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftExecC2C (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Measure execution time
    hipDeviceSynchronize();
    // Allocate CUDA events
    hipEvent_t start;
    checkCudaErrors(hipEventCreate(&start));
    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&stop));
    // Record the start event
    checkCudaErrors(hipEventRecord(start, NULL));
    // Repeatedly execute cuFFT
    int nIter = 300;
    for (int i = 0; i < nIter; i++){
        result = hipfftXtExec(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                              reinterpret_cast<hipfftComplex *>(d_odata), \
                              HIPFFT_FORWARD);
    }
    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, NULL));
    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));
    // Calculate performance
    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
    float msecPerFFT = msecTotal / nIter;

    // Copy Device memory to host
    Csingle *h_odata = (Csingle *)malloc(mem_size);
    checkCudaErrors(hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost));

    // Print result
    if (DISPLAY_DATA == 1) {
        printf("FFT result: \n");
        for (unsigned int i = 0; i < input_size; i++) {
            for (unsigned int j = 0; j < input_size; j++) {
                printf("x[%d, %d]=(%.2f, %.2f); \n", j, i, h_odata[i*input_size + j].x, h_odata[i*input_size + j].y);
            }
            printf("\n");
        }
    }

    // Print the performance
    printf("Performance of cuFFT1D: Problem size= %d * %d, Time= %.5f msec\n", \
        input_size, input_size,
        msecPerFFT);

    // Clean up content and memory
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    hipfftDestroy(plan);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));
    free(h_idata);
    free(h_odata);

    return 0;
}

int run_test_2D(int input_size){
    printf("[cuFFT 2D] is starting...\n");
    int mem_size = input_size * input_size * sizeof(Csingle);
    Csingle *h_idata = (Csingle *)malloc(mem_size);
    
    // Intialize the memory for the input data
    for (unsigned int i = 0; i < input_size; i++) {    
        for (unsigned int j = 0; j < input_size; j++) {
            h_idata[i*input_size + j].x = rand() / (0.5 * static_cast<float>(RAND_MAX)) - 1;
            h_idata[i*input_size + j].y = rand() / (0.5 * static_cast<float>(RAND_MAX)) - 1;
        }
        if (input_size == 4) {
            h_idata[i*input_size + 0].x = 1; h_idata[i*input_size + 0].y = 2;
            h_idata[i*input_size + 1].x = 0; h_idata[i*input_size + 1].y = 0; 
            h_idata[i*input_size + 2].x = 0; h_idata[i*input_size + 2].y = 1;
            h_idata[i*input_size + 3].x = -1; h_idata[i*input_size + 3].y = 0;
        }
    }
    if (DISPLAY_DATA == 1) {
        printf("Input data: \n");
        for (unsigned int i = 0; i < input_size; i++) {    
            for (unsigned int j = 0; j < input_size; j++) {
                printf("x[%d, %d]=(%.2f, %.2f); \n", j, i, h_idata[i*input_size + j].x, h_idata[i*input_size + j].y);
            }
            printf("\n"); 
        }
    }
    // Allocate device momory for input and output
    Csingle *d_idata, *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // Copy host data to device
    checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

    // cuFFT plan
    hipfftResult result;
    hipfftHandle plan;
    size_t workSize;
    long long int input_size_long[2];
    result = hipfftCreate(&plan);
    input_size_long[0] = input_size;
    input_size_long[1] = input_size;
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftCreate (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    result = hipfftXtMakePlanMany(plan, 2, input_size_long, NULL, 1, 1, \
                         HIP_C_32F, NULL, 1, 1, HIP_C_32F, 1, \
                         &workSize, HIP_C_32F);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftXtMakePlanMany (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    printf("Temporary buffer size %li bytes\n", workSize);

    // cuFFT warm-up execution
    result = hipfftXtExec(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                          reinterpret_cast<hipfftComplex *>(d_odata), \
                          HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftExecC2C (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Measure execution time
    hipDeviceSynchronize();
    // Allocate CUDA events
    hipEvent_t start;
    checkCudaErrors(hipEventCreate(&start));
    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&stop));
    // Record the start event
    checkCudaErrors(hipEventRecord(start, NULL));
    // Repeatedly execute cuFFT
    int nIter = 300;
    for (int i = 0; i < nIter; i++){
        result = hipfftXtExec(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                              reinterpret_cast<hipfftComplex *>(d_odata), \
                              HIPFFT_FORWARD);
    }
    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, NULL));
    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));
    // Calculate performance
    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
    float msecPerFFT = msecTotal / nIter;

    // Copy Device memory to host
    Csingle *h_odata = (Csingle *)malloc(mem_size);
    checkCudaErrors(hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost));

    // Print result
    if (DISPLAY_DATA == 1) {
        printf("FFT result: \n");
        for (unsigned int i = 0; i < input_size; i++) {
            for (unsigned int j = 0; j < input_size; j++) {
                printf("x[%d, %d]=(%.2f, %.2f); \n", j, i, h_odata[i*input_size + j].x, h_odata[i*input_size + j].y);
            }
            printf("\n");
        }
    }

    // Print the performance
    printf("Performance of cuFFT2D: Problem size= %d * %d, Time= %.5f msec\n", \
        input_size, input_size,
        msecPerFFT);

    // Clean up content and memory
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    hipfftDestroy(plan);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));
    free(h_idata);
    free(h_odata);

    return 0;
}



int main(int argc, char **argv)
{
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
            checkCmdLineFlag(argc, (const char **)argv, "?") ||
            checkCmdLineFlag(argc, (const char **)argv, "h")) {
        printf("Usage: -n=size (Input vector size)"
	       " -device=ID (ID > 0 for deviceID)"
               " -display=show_result (0 or 1) \n"); 
        exit(EXIT_SUCCESS);
    }
    
    // Set block size
    int block_size = 32;
 
    // Device ID by defualt is 0
    int devID = 0;
    if (checkCmdLineFlag(argc, (const char **)argv, "device")) {
        devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(devID);
    }
    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);
    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    error = hipGetDeviceProperties(&deviceProp, devID);
    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }
    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    // Input size by defualt is 8
    int n = block_size * 8;
    if (checkCmdLineFlag(argc, (const char **)argv, "n")) {
        n = getCmdLineArgumentInt(argc, (const char **)argv, "n");
    }
     
    // Set display mode
    if (checkCmdLineFlag(argc, (const char **)argv, "display")) {
        int entered_mode = getCmdLineArgumentInt(argc, (const char **)argv, "display");
        if (entered_mode == 0)  DISPLAY_DATA = 0;
    }

    printf("Problem size = %d * %d\n", n, n);

    printf("[Testing of cuFFT 1D and 2D] - Starting...\n");
    
    int test1D = run_test_1D(n);
    int test2D = run_test_2D(n);

    exit(test1D || test2D);
} 
