#include "hip/hip_runtime.h"
/* 
 * A program that compare performance of gfft and cuFFT library
 * Test the speed and accuracy of FP16 and FP32 calculation
 */

// C library, CUDA runtime, helpers, and utilities
#include "../util/my_include.h"
#include <vector>

// gfft
#include "../util/gfft_using_fft4.h"

// CUFFT
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

typedef half2 Chalf;
typedef float2 Csingle;

const float NORM = 1.0f;
const int BATCH = 16;
const int SIZE = 256;
const int DISPLAY_DATA = 0;


int cuFFT32(int N, int B, Csingle* X, Csingle* FX){
    // Allocate unified momory for input and output
    int mem_size = N * B *sizeof(Csingle);
    Csingle *d_idata, *d_odata;
    checkCudaErrors(hipMallocManaged((void **) &d_idata, mem_size));
    checkCudaErrors(hipMallocManaged((void **) &d_odata, mem_size));

    // Copy input data to memory
    checkCudaErrors(hipMemcpy(d_idata, X, mem_size, hipMemcpyHostToDevice));

    // cuFFT plan
    hipfftResult result;
    hipfftHandle plan;
    size_t workSize;
    long long int input_size_long = N;
    result = hipfftCreate(&plan);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftCreate (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    result = hipfftXtMakePlanMany(plan, 1, &input_size_long, NULL, 1, 1, \
                         HIP_C_32F, NULL, 1, 1, HIP_C_32F, B, \
                         &workSize, HIP_C_32F);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftXtMakePlanMany (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }

    // cuFFT execution
    result = hipfftXtExec(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                          reinterpret_cast<hipfftComplex *>(d_odata), \
                          HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftExecC2C (execution) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Copy Device memory to output
    checkCudaErrors(hipMemcpy(FX, d_odata, mem_size, hipMemcpyDeviceToHost));

    // Clean up content and memory
    hipfftDestroy(plan);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    return 0;
}


int main(int argc, char **argv)
{
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
            checkCmdLineFlag(argc, (const char **)argv, "?") ||
            checkCmdLineFlag(argc, (const char **)argv, "h")) {
        printf("Usage: -norm=upper_bound (Max norm of input elements)\n"
               " -n=size (Input vector size)\n"
               " -batch=batch_size (Number of input vectors)\n"
               " -display=show_result (0 or 1) \n"); 
        exit(EXIT_SUCCESS);
    }
 

    // Get and set parameter 
    //// Norm
    float norm = NORM;
    if (checkCmdLineFlag(argc, (const char **)argv, "norm")) {
        norm = getCmdLineArgumentInt(argc, (const char **)argv, "norm");
    }

    //// Input size
    int n = SIZE;
    if (checkCmdLineFlag(argc, (const char **)argv, "n")) {
        n = getCmdLineArgumentInt(argc, (const char **)argv, "n");
    }

    //// Batch size
    int batch = BATCH;
    if (checkCmdLineFlag(argc, (const char **)argv, "batch")) {
        batch = getCmdLineArgumentInt(argc, (const char **)argv, "batch");
    }
    
    //// Result display mode
    int display = DISPLAY_DATA;
    if (checkCmdLineFlag(argc, (const char **)argv, "display")) {
        display = getCmdLineArgumentInt(argc, (const char **)argv, "display");
    }
    
    // Start program
    printf("Problem size = %d, batch size = %d, norm = %f\n", n, batch, norm);

    printf("[Testing of cuFFT FP32] - Starting...\n");

    // Define input and output
    Csingle X_32[n * batch], FX_32[n * batch];
   
    // Run experiment
    for (int i = 0; i < 1; i++){
        // Initialize input
        srand(time(NULL));
        for (int j = 0; j < n * batch; j++){
            X_32[j].x = (float)rand() / (float)(RAND_MAX) * 2 * norm - norm;
            X_32[j].y = (float)rand() / (float)(RAND_MAX) * 2 * norm - norm;
            if (display == 1){
                printf("X[%d] = (%.10f, %.10f) \n", j, X_32[j].x, X_32[j].y);
            }
  
        }
        // Call cuFFT32
        cuFFT32(n, batch, X_32, FX_32);
    }

    exit(0);
}
