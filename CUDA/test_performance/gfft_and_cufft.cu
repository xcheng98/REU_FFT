#include "hip/hip_runtime.h"
/* 
 * A program that compare performance of gfft and cuFFT library
 * Test the speed and accuracy of FP16 and FP32 calculation
 */

// C library, CUDA runtime, helpers, and utilities
#include "../util/my_include.h"
#include <vector>

// gfft
#include "../util/gfft_using_fft4.h"

// CUFFT
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

typedef half2 Chalf;
typedef float2 Csingle;

const float NORM = 1.0f;
const int BATCH = 16;
const int SIZE = 256;
const int BLOCK_SIZE = 32;
const int DISPLAY_DATA = 0;
const int DEVICE = 0;

#define __START__ hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start, 0);
#define __STOP__(_V) hipEventRecord(stop, 0); hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop); _V.push_back(duration); hipEventDestroy(start); hipEventDestroy(stop);

float show_mean(std::vector<float> v)
{
    float sum = 0;
    for (int i = 0; i < v.size(); i++)
        sum += v[i];
    return sum / v.size(); 
}

int cuFFT32(int N, int B, Csingle* X, Csingle* FX){
    // Allocate unified momory for input and output
    int mem_size = N * B *sizeof(Csingle);
    Csingle *d_idata, *d_odata;
    checkCudaErrors(hipMallocManaged((void **) &d_idata, mem_size));
    checkCudaErrors(hipMallocManaged((void **) &d_odata, mem_size));

    // Copy input data to memory
    checkCudaErrors(hipMemcpy(d_idata, X, mem_size, hipMemcpyHostToDevice));

    // cuFFT plan
    hipfftResult result;
    hipfftHandle plan;
    size_t workSize;
    long long int input_size_long = N;
    result = hipfftCreate(&plan);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftCreate (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    result = hipfftXtMakePlanMany(plan, 1, &input_size_long, NULL, 1, 1, \
                         HIP_C_32F, NULL, 1, 1, HIP_C_32F, B, \
                         &workSize, HIP_C_32F);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftXtMakePlanMany (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }

    // cuFFT execution
    result = hipfftXtExec(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                          reinterpret_cast<hipfftComplex *>(d_odata), \
                          HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftExecC2C (execution) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Copy Device memory to output
    checkCudaErrors(hipMemcpy(FX, d_odata, mem_size, hipMemcpyDeviceToHost));

    // Clean up content and memory
    hipfftDestroy(plan);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    return 0;
}

int cuFFT16(int N, int B, Chalf* X, Chalf* FX){
    // Allocate unified momory for input and output
    int mem_size = N * B *sizeof(Chalf);
    Chalf *d_idata, *d_odata;
    checkCudaErrors(hipMallocManaged((void **) &d_idata, mem_size));
    checkCudaErrors(hipMallocManaged((void **) &d_odata, mem_size));

    // Copy input data to memory
    checkCudaErrors(hipMemcpy(d_idata, X, mem_size, hipMemcpyHostToDevice));

    // cuFFT plan
    hipfftResult result;
    hipfftHandle plan;
    size_t workSize;
    long long int input_size_long = N;
    result = hipfftCreate(&plan);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftCreate (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    result = hipfftXtMakePlanMany(plan, 1, &input_size_long, NULL, 1, 1, \
                         HIP_C_16F, NULL, 1, 1, HIP_C_16F, B, \
                         &workSize, HIP_C_16F);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftXtMakePlanMany (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }

    // cuFFT execution
    result = hipfftXtExec(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                          reinterpret_cast<hipfftComplex *>(d_odata), \
                          HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftExecC2C (execution) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Copy Device memory to output
    checkCudaErrors(hipMemcpy(FX, d_odata, mem_size, hipMemcpyDeviceToHost));

    // Clean up content and memory
    hipfftDestroy(plan);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    return 0;
}

int main(int argc, char **argv)
{
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
            checkCmdLineFlag(argc, (const char **)argv, "?") ||
            checkCmdLineFlag(argc, (const char **)argv, "h")) {
        printf("Usage: -norm=upper_bound (Max norm of input elements)"
               " -n=size (Input vector size)\n"
               " -batch=batch_size (Number of input vectors)\n"
               " -bs=block_size (Number of threads in a block)\n"
               " -display=show_result (0 or 1) \n" 
	       " -device=ID (ID >= 0 for deviceID)\n");
        exit(EXIT_SUCCESS);
    }
 

    // Get and set parameter 
    //// Norm
    float norm = NORM;
    if (checkCmdLineFlag(argc, (const char **)argv, "norm")) {
        norm = getCmdLineArgumentInt(argc, (const char **)argv, "norm");
    }

    //// Input size
    int n = SIZE;
    if (checkCmdLineFlag(argc, (const char **)argv, "n")) {
        n = getCmdLineArgumentInt(argc, (const char **)argv, "n");
    }

    //// Batch size
    int batch = BATCH;
    if (checkCmdLineFlag(argc, (const char **)argv, "batch")) {
        batch = getCmdLineArgumentInt(argc, (const char **)argv, "batch");
    }
    
    //// Block size
    int bs = BLOCK_SIZE;
    if (checkCmdLineFlag(argc, (const char **)argv, "bs")) {
        bs = getCmdLineArgumentInt(argc, (const char **)argv, "bs");
    }
    
    //// Result display mode
    int display = DISPLAY_DATA;
    if (checkCmdLineFlag(argc, (const char **)argv, "display")) {
        display = getCmdLineArgumentInt(argc, (const char **)argv, "display");
    }
    
    //// Device ID by defualt is 0
    int device = DEVICE;
    if (checkCmdLineFlag(argc, (const char **)argv, "device")) {
        device = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(device);
    }
    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&device);
    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    error = hipGetDeviceProperties(&deviceProp, device);
    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }
    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", device, deviceProp.name, deviceProp.major, deviceProp.minor);
    }


    // Start program
    printf("Problem size = %d, batch size = %d, norm = %f\n", n, batch, norm);

    printf("[Testing of gfft and cuFFT] - Starting...\n");

    // Define event, result data structure
    hipEvent_t start, stop;
    std::vector<float> cuFFT32Run, cuFFT16Run, gfftRun;
    std::vector<float> cuFFT16Error, gfftError;
    float duration, error1, error2;

    // Define input and output
    float X_re[n * batch], X_im[n * batch], FX_re[n * batch], FX_im[n * batch];
    Csingle X_32[n * batch], FX_32[n * batch];
    Chalf X_16[n * batch], FX_16[n * batch];
   
    // Run experiment
    for (int i = 0; i < 10; i++){
        // Initialize input
        srand(time(NULL));
        for (int j = 0; j < n * batch; j++){
            X_re[j] = (float)rand() / (float)(RAND_MAX) * 2 * norm - norm;
            X_im[j] = (float)rand() / (float)(RAND_MAX) * 2 * norm - norm;
            X_32[j].x = X_re[j]; X_32[j].y = X_im[j];
            X_16[j].x = (half)X_re[j]; X_16[j].y = (half)X_im[j];
            if (display == 1){
                printf("X[%d] = (%.10f, %.10f) \n", j, X_re[j], X_im[j]);
            }
  
        }
        // Call cuFFT32
        __START__
        cuFFT32(n, batch, X_32, FX_32);
        __STOP__(cuFFT32Run)


        // Call cuFFT16
        __START__
        cuFFT16(n, batch, X_16, FX_16);
        __STOP__(cuFFT16Run)


        // Call gfft
        __START__
        gfft(n, batch, X_re, X_im, FX_re, FX_im);
        __STOP__(gfftRun)

        // Calculate error
        for (int j = 0; j < n * batch; j++){
            error1 += (float)fabs((float)(FX_16[j].x) - FX_32[j].x);
            error1 += (float)fabs((float)(FX_16[j].y) - FX_32[j].y);
            error2 += (float)fabs(FX_re[j] - FX_32[j].x);
            error2 += (float)fabs(FX_im[j] - FX_32[j].y);
        }
        cuFFT16Error.push_back(error1 / (n * batch));
        gfftError.push_back(error2 / (n * batch));
    }

    printf("Time of cuFFT32: %f milliseconds\n", show_mean(cuFFT32Run)); 
    printf("Time of cuFFT16: %f milliseconds, error = %.10f\n", show_mean(cuFFT16Run), show_mean(cuFFT16Error)/norm); 
    printf("Time of gfft: %f milliseconds, error = %.10f\n", show_mean(gfftRun), show_mean(gfftError)/norm); 

    exit(0);
} 
