#include "hip/hip_runtime.h"
/* 
 * A program that compare acceleration of gemm, cufft32, cufft16
 */

// C includes
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <vector>

// CUDA includes
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
// CUFFT
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

// nvidia helper
#include "../checkCudaErrors.h"
#include "../helper_string.h"

typedef half2 Chalf;
typedef float2 Csingle;

const float NORM = 1.0f;
const int BATCH = 16;
const int SIZE = 256;
const int ITERATION = 10;
const int DISPLAY_DATA = 0;
const int DEVICE = 0;

#define __START__ hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start, 0);
#define __STOP__(_V) hipEventRecord(stop, 0); hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop); _V.push_back(duration); hipEventDestroy(start); hipEventDestroy(stop);


hipEvent_t start, stop;
std::vector<float> cuFFT32Run, cuFFT16Run, gemmRun;
float duration;


float show_mean(std::vector<float> v)
{
    float sum = 0;
    for (int i = 0; i < v.size(); i++)
        sum += v[i];
    return sum / v.size(); 
}

int cuFFT32(int N, Csingle* X, Csingle* FX, int B){
    // Allocate unified momory for input and output
    int mem_size = N * N * B *sizeof(Csingle);
    Csingle *d_idata, *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // Copy input data to memory
    checkCudaErrors(hipMemcpy(d_idata, X, mem_size, hipMemcpyHostToDevice));

    // cuFFT plan
    hipfftResult result;
    hipfftHandle plan;
    size_t workSize;
    long long int input_size_long[2] = {N, N};
    result = hipfftCreate(&plan);
    if (result != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "In cuFFT32: hipfftCreate plan returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    result = hipfftXtMakePlanMany(plan, 2, input_size_long, NULL, 1, 1, \
                         HIP_C_32F, NULL, 1, 1, HIP_C_32F, B, \
                         &workSize, HIP_C_32F);
    if (result != HIPFFT_SUCCESS)
    {
        printf("In cuFFT32: hipfftXtMakePlanMany returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }


    __START__
    // cuFFT execution
    result = hipfftXtExec(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                          reinterpret_cast<hipfftComplex *>(d_odata), \
                          HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS)
    {
        printf("In cuFFT32: hipfftExecC2C (execution) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    __STOP__(cuFFT32Run)

    // Copy Device memory to output
    checkCudaErrors(hipMemcpy(FX, d_odata, mem_size, hipMemcpyDeviceToHost));

    // Clean up content and memory
    hipfftDestroy(plan);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    return 0;
}

int cuFFT16(int N, Chalf* X, Chalf* FX, int B){
    // Allocate unified momory for input and output
    int mem_size = N * N * B *sizeof(Chalf);
    Chalf *d_idata, *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // Copy input data to memory
    checkCudaErrors(hipMemcpy(d_idata, X, mem_size, hipMemcpyHostToDevice));

    // cuFFT plan
    hipfftResult result;
    hipfftHandle plan;
    size_t workSize;
    long long int input_size_long[2] = {N, N};
    result = hipfftCreate(&plan);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftCreate (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    result = hipfftXtMakePlanMany(plan, 2, input_size_long, NULL, 1, 1, \
                         HIP_C_16F, NULL, 1, 1, HIP_C_16F, B, \
                         &workSize, HIP_C_16F);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftXtMakePlanMany (plan) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }

    __START__
    // cuFFT execution
    result = hipfftXtExec(plan, reinterpret_cast<hipfftComplex *>(d_idata), \
                          reinterpret_cast<hipfftComplex *>(d_odata), \
                          HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS)
    {
        printf("hipfftExecC2C (execution) returned error code %d, line(%d)\n", result, __LINE__);
        exit(EXIT_FAILURE);
    }
    __STOP__(cuFFT16Run)

    // Copy Device memory to output
    checkCudaErrors(hipMemcpy(FX, d_odata, mem_size, hipMemcpyDeviceToHost));

    // Clean up content and memory
    hipfftDestroy(plan);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    return 0;
}

int gemm(int N, half* X, half* FX, int B){
    // Allocate unified momory for input and output
    int mem_size = N * N * B *sizeof(half);
    Chalf *d_idata, *d_idata2, *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    checkCudaErrors(hipMalloc((void **) &d_idata2, mem_size));
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // Copy input data to memory
    checkCudaErrors(hipMemcpy(d_idata, X, mem_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_idata2, X, mem_size, hipMemcpyHostToDevice));

    // cublas
    hipblasStatus_t status;
    hipblasHandle_t handle;
    half alpha = 1.0f, beta = 0.0f;
    // Initialize cublas with global cublas handle and status
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!!! CUBLAS initialization error.\n");
        exit(1);
    }
    // Allow cublas to use Tensor Core
    status = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH); 
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!!! CUBLAS setting math mode error.\n");
        exit(1);
    }
    
    __START__
    status = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, 
        d_idata, HIP_R_16F, N, d_idata2, HIP_R_16F, N, &beta,
        d_odata, HIP_R_16F, N, HIP_R_16F, HIPBLAS_GEMM_DEFAULT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!!! CUBLAS kernel execution error .\n");
        exit(-1);
    }
    __STOP__(gemmRun)

    // Copy Device memory to output
    checkCudaErrors(hipMemcpy(FX, d_odata, mem_size, hipMemcpyDeviceToHost));

    // Clean up content and memory
    status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!!! CUBLAS shutdown error.\n");
        exit(1);
    }
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    return 0;
}


int get_parameters(int argc, char **argv, int& help_info, float& norm, int& n, int& batch, int& iter, int& display, int& device){
    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
            checkCmdLineFlag(argc, (const char **)argv, "?") ||
            checkCmdLineFlag(argc, (const char **)argv, "h")) {
        printf("Usage: -norm=upper_bound (Max norm of input elements)\n"
               " -n=size (Input vector size)\n"
               " -batch=batch_size (Number of input vectors)\n"
               " -iter=iteration (Times of experiments)\n"
               " -display=show_result (0 or 1) \n" 
               " -device=ID (ID >= 0 for deviceID)\n");
        help_info = 1;
        return 0;
    }

    // Get and set parameter 
    if (checkCmdLineFlag(argc, (const char **)argv, "norm")) {
        norm = getCmdLineArgumentFloat(argc, (const char **)argv, "norm");
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "n")) {
        n = getCmdLineArgumentInt(argc, (const char **)argv, "n");
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "batch")) {
        batch = getCmdLineArgumentInt(argc, (const char **)argv, "batch");
    }
    
    if (checkCmdLineFlag(argc, (const char **)argv, "iter")) {
        iter = getCmdLineArgumentInt(argc, (const char **)argv, "iter");
    }
    
    if (checkCmdLineFlag(argc, (const char **)argv, "display")) {
        display = getCmdLineArgumentInt(argc, (const char **)argv, "display");
    }
    
    if (checkCmdLineFlag(argc, (const char **)argv, "device")) {
        device = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(device);
    }

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&device);
    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    error = hipGetDeviceProperties(&deviceProp, device);
    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }
    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", device, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    return 0;
}


int main(int argc, char **argv)
{
    int help_info = 0;
    float norm = NORM;
    int n = SIZE;
    int batch = BATCH;
    int iter = ITERATION;
    int display = DISPLAY_DATA;
    int device = DEVICE;

    get_parameters(argc, argv, help_info, norm, n, batch, iter, display, device);

    if (help_info == 1){
        exit(EXIT_SUCCESS);
    }

    // Start program
    printf("Problem size = %d, batch size = %d, norm = %f, iteration = %d\n", n, batch, norm, iter);

    printf("[Testing acceleration] - Starting...\n");

    // Define and zero initialize input and output
    half* X_re = new half[n * n * batch]();
    half* FX_re = new half[n * n * batch]();
    Csingle* X_32 = new Csingle[n * n * batch]();
    Csingle* FX_32 = new Csingle[n * n * batch]();
    Chalf* X_16 = new Chalf[n * n * batch]();
    Chalf* FX_16 = new Chalf[n * n * batch]();

    // Warm up
    cuFFT32(n, X_32, FX_32, batch);
    cuFFT16(n, X_16, FX_16, batch);
    gemm(n, X_re, FX_re, batch);

    cuFFT32Run.pop_back();
    cuFFT16Run.pop_back();
    gemmRun.pop_back();

    printf("Warm up completed, start experiments...\n");
 
    // Run experiment
    for (int i = 0; i < iter; i++){
        // Initialize input
        srand(time(NULL));
        for (int j = 0; j < n * batch; j++){
            X_re[j] = (float)rand() / (float)(RAND_MAX) * 2 * norm - norm;
            FX_re[j] = (float)rand() / (float)(RAND_MAX) * 2 * norm - norm;
            X_32[j].x = X_re[j]; X_32[j].y = FX_re[j];
            X_16[j].x = (half)X_re[j]; X_16[j].y = (half)FX_re[j];
        }

        cuFFT32(n, X_32, FX_32, batch);
        cuFFT16(n, X_16, FX_16, batch);
        gemm(n, X_re, FX_re, batch);
    }

    // Print experiment result
    printf("Time of cuFFT32: %f milliseconds\n", show_mean(cuFFT32Run)); 
    printf("Time of cuFFT16: %f milliseconds\n", show_mean(cuFFT16Run)); 
    printf("Time of gemm: %f milliseconds\n", show_mean(gemmRun)); 

    // Free input and output memory
    delete [] X_re;
    delete [] FX_re;
    delete [] X_32;
    delete [] FX_32;
    delete [] X_16;
    delete [] FX_16;

    exit(0);
} 
